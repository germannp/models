#include "hip/hip_runtime.h"
// Simulate elongation of Xenopus aggregates, see Green (2014) Dev Dyn.
#include <math.h>
#include <stdio.h>
#include <thread>
#include <functional>
#include <hiprand/hiprand_kernel.h>

#include "../include/cudebug.cuh"
#include "../include/dtypes.cuh"
#include "../include/solvers.cuh"
#include "../include/inits.cuh"
#include "../include/links.cuh"
#include "../include/polarity.cuh"
#include "../include/vtk.cuh"


const auto n_cells = 500;
const auto r_protrusion = 1.5f;
const auto prots_per_cell = 1;
const auto n_time_steps = 500;
const auto dt = 0.2f;


__device__ Po_cell lb_force(Po_cell Xi, Po_cell r, float dist, int i, int j) {
    Po_cell dF {0};
    if (i == j) return dF;

    if (dist > 1) return dF;

    auto F = fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2;
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    return dF;
}


__device__ void protrusion_force(const Po_cell* __restrict__ d_X, const int a, const int b,
        const float strength, Po_cell* d_dX) {
    auto r = d_X[a] - d_X[b];
    auto dist = norm3df(r.x, r.y, r.z);

    atomicAdd(&d_dX[a].x, -strength*r.x/dist);
    atomicAdd(&d_dX[a].y, -strength*r.y/dist);
    atomicAdd(&d_dX[a].z, -strength*r.z/dist);
    atomicAdd(&d_dX[b].x, strength*r.x/dist);
    atomicAdd(&d_dX[b].y, strength*r.y/dist);
    atomicAdd(&d_dX[b].z, strength*r.z/dist);

    Polarity r_hat {acosf(-r.z/dist), atan2(-r.y, -r.x)};
    auto Fa = pcp_force(d_X[a], r_hat);
    atomicAdd(&d_dX[a].theta, strength*Fa.theta);
    atomicAdd(&d_dX[a].phi, strength*Fa.phi);

    // r_hat.theta = acosf(r.z/dist);
    // r_hat.phi = atan2(r.y, r.x);
    auto Fb = pcp_force(d_X[b], r_hat);
    atomicAdd(&d_dX[b].theta, strength*Fb.theta);
    atomicAdd(&d_dX[b].phi, strength*Fb.phi);
}


__global__ void update_protrusions(const Grid<n_cells>* __restrict__ d_grid,
        const Po_cell* __restrict d_X, Link* d_link, hiprandState* d_state) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= n_cells*prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5)/prots_per_cell);
    auto rand_nb_cube = d_grid->d_cube_id[j]
        + d_moore_nhood[min(static_cast<int>(hiprand_uniform(&d_state[i])*27), 26)];
    auto cells_in_cube = d_grid->d_cube_end[rand_nb_cube] - d_grid->d_cube_start[rand_nb_cube];
    if (cells_in_cube < 1) return;

    auto a = d_grid->d_point_id[j];
    auto b = d_grid->d_point_id[d_grid->d_cube_start[rand_nb_cube]
        + min(static_cast<int>(hiprand_uniform(&d_state[i])*cells_in_cube), cells_in_cube - 1)];
    D_ASSERT(a >= 0); D_ASSERT(a < n_cells);
    D_ASSERT(b >= 0); D_ASSERT(b < n_cells);
    if (a == b) return;

    auto new_r = d_X[a] - d_X[b];
    auto new_dist = norm3df(new_r.x, new_r.y, new_r.z);
    if (new_dist > r_protrusion) return;

    auto link = &d_link[a*prots_per_cell + i%prots_per_cell];
    auto not_initialized = link->a == link->b;
    auto old_r = d_X[link->a] - d_X[link->b];
    auto old_dist = norm3df(old_r.x, old_r.y, old_r.z);
    Polarity old_rhat {acosf(-old_r.z/old_dist), atan2(-old_r.y, -old_r.x)};
    auto old_pcp = pol_scalar_product(d_X[a], old_rhat);
    Polarity new_rhat {acosf(-new_r.z/new_dist), atan2(-new_r.y, -new_r.x)};
    auto new_pcp = pol_scalar_product(d_X[a], new_rhat);
    auto noise = hiprand_uniform(&d_state[i])*0;
    auto more_along_pcp = fabs(new_pcp) > fabs(old_pcp)*(1.f - noise);
    if (not_initialized or more_along_pcp) {
        link->a = a;
        link->b = b;
    }
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<Po_cell, n_cells, Grid_solver> bolls;
    uniform_sphere(0.733333, bolls);
    for (auto i = 0; i < n_cells; i++) {
        bolls.h_X[i].y /= 5;
        bolls.h_X[i].theta = acos(2.*rand()/(RAND_MAX + 1.) - 1.);
        bolls.h_X[i].phi = 2.*M_PI*rand()/(RAND_MAX + 1.);
    }
    bolls.copy_to_device();
    Links<static_cast<int>(n_cells*prots_per_cell)> protrusions;
    auto intercalation = std::bind(
        link_forces<static_cast<int>(n_cells*prots_per_cell), Po_cell, protrusion_force>,
        protrusions, std::placeholders::_1, std::placeholders::_2);

    // Simulate elongation
    Vtk_output output("aggregate");
    Grid<n_cells> grid;
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        bolls.copy_to_host();
        protrusions.copy_to_host();

        grid.build(bolls, r_protrusion);
        update_protrusions<<<(protrusions.get_d_n() + 32 - 1)/32, 32>>>(grid.d_grid,
            bolls.d_X, protrusions.d_link, protrusions.d_state);
        bolls.take_step<lb_force>(dt, intercalation);

        output.write_positions(bolls);
        output.write_links(protrusions);
        output.write_polarity(bolls);
    }

    return 0;
}
