#include "hip/hip_runtime.h"
// Simulate elongation of semisphere
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/protrusions.cuh"
#include "../lib/epithelium.cuh"
#include "../lib/vtk.cuh"


const auto R_MAX = 1;
const auto R_MIN = 0.6;
const auto N_MAX = 61000;
const auto R_LINK = 1.5;
const auto LINKS_P_CELL = 1.f;
const auto N_TIME_STEPS = 500;
const auto DELTA_T = 0.2;
enum CELL_TYPES {MESENCHYME, STRETCHED_EPI, EPITHELIUM};

__device__ __managed__ auto n_cells = 5000;
__device__ __managed__ CELL_TYPES cell_type[N_MAX];
__device__ __managed__ Protrusions<static_cast<int>(N_MAX*LINKS_P_CELL)> prots;


MAKE_DTYPE(lbcell, x, y, z, w, phi, theta);

__device__ __managed__ Solution<lbcell, N_MAX, LatticeSolver> X;


__device__ lbcell cubic_w_diffusion(lbcell Xi, lbcell Xj, int i, int j) {
    lbcell dF {0};
    if (i == j) {
        assert(Xi.w >= 0);
        dF.w = (cell_type[i] > MESENCHYME) - 0.01*Xi.w;
        return dF;
    }

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    auto F = 2*(R_MIN - dist)*(R_MAX - dist) + (R_MAX - dist)*(R_MAX - dist);
    dF.x = r.x*F/dist*(Xi.x > 0);
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    auto D = dist < R_MAX ? 0.1 : 0;
    dF.w = - r.w*D;

    if (cell_type[i] == MESENCHYME or cell_type[j] == MESENCHYME) return dF;

    if (dist < 0.733333) cell_type[i] = EPITHELIUM;
    dF += polarity_force(Xi, Xj)*0.2;
    return dF;
}

__device__ __managed__ auto d_potential = cubic_w_diffusion;


__device__ lbcell count_neighbours(lbcell Xi, lbcell Xj, int i, int j) {
    lbcell dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    dF.w = dist < R_MAX ? 1 : 0;
    return dF;
}

__device__ __managed__ auto d_count = count_neighbours;


__global__ void update_links(const int* __restrict__ cell_id,
        const int* __restrict__ cube_id, const int* __restrict__ cube_start,
        const int* __restrict__ cube_end) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= n_cells*LINKS_P_CELL) return;

    auto j = static_cast<int>(hiprand_uniform(&prots.rand_states[i])*n_cells);
    auto rand_cube = cube_id[j]
        +  static_cast<int>(hiprand_uniform(&prots.rand_states[i])*3) - 1
        + (static_cast<int>(hiprand_uniform(&prots.rand_states[i])*3) - 1)*LATTICE_SIZE
        + (static_cast<int>(hiprand_uniform(&prots.rand_states[i])*3) - 1)*LATTICE_SIZE*LATTICE_SIZE;
    auto cells_in_cube = cube_end[rand_cube] - cube_start[rand_cube];
    if (cells_in_cube < 1) return;

    auto k = cube_start[rand_cube]
        + static_cast<int>(hiprand_uniform(&prots.rand_states[i])*cells_in_cube);
    auto r = X[cell_id[j]] - X[cell_id[k]];
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if ((j != k) and (cell_type[cell_id[j]] == MESENCHYME)
            and (cell_type[cell_id[k]] == MESENCHYME)
            and (dist < R_LINK) and (fabs(r.w/(X[cell_id[j]].w + X[cell_id[k]].w)) > 0.2)) {
            // and (fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
        prots.links[i][0] = cell_id[j];
        prots.links[i][1] = cell_id[k];
    }
}

void intercalation(const lbcell* __restrict__ X, lbcell* dX) {
    intercalate<<<(n_cells*LINKS_P_CELL + 32 - 1)/32, 32>>>(X, dX, prots, 0.5,
        n_cells*LINKS_P_CELL);
    hipDeviceSynchronize();
}


__global__ void proliferate(float rate, float mean_distance) {
    assert(rate*n_cells <= N_MAX);
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= n_cells) return;

    if (cell_type[i] == EPITHELIUM) {
        cell_type[i] = STRETCHED_EPI;
        return;
    }

    if (cell_type[i] == MESENCHYME) {
        auto r = hiprand_uniform(&prots.rand_states[i]);
        if (r > rate) return;
    }

    auto n = atomicAdd(&n_cells, 1);
    auto phi = hiprand_uniform(&prots.rand_states[i])*M_PI;
    auto theta = hiprand_uniform(&prots.rand_states[i])*2*M_PI;
    X[n].x = X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    X[n].y = X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    X[n].z = X[i].z + mean_distance/4*cosf(theta);
    X[n].w = X[i].w/2;
    X[i].w = X[i].w/2;
    X[n].phi = X[i].phi;
    X[n].theta = X[i].theta;
    cell_type[n] = cell_type[i] == MESENCHYME ? MESENCHYME : STRETCHED_EPI;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(0.733333, X, n_cells);
    for (auto i = 0; i < n_cells; i++) {
        X[i].x = fabs(X[i].x);
        X[i].y = X[i].y/1.5;
        X[i].w = 0;
        cell_type[i] = MESENCHYME;
    }
    init_protrusions(prots);

    // Relax
    VtkOutput relax_output("relaxation");
    for (auto time_step = 0; time_step <= 200; time_step++) {
        X.step(DELTA_T, d_potential, n_cells);
        relax_output.print_progress();
    }
    relax_output.print_done();

    // Find epithelium
    X.step(1, d_count, n_cells);
    // X.z_order(n_cells, 2.);
    for (auto i = 0; i < n_cells; i++) {
        if (X[i].w < 12 and X[i].x > 0) {
            cell_type[i] = STRETCHED_EPI;
            auto dist = sqrtf(X[i].x*X[i].x + X[i].y*X[i].y + X[i].z*X[i].z);
            X[i].phi = atan2(X[i].y, X[i].x);
            X[i].theta = acosf(X[i].z/dist);
        } else {
            X[i].phi = 0;
            X[i].theta = 0;
        }
        X[i].w = 0;
    }

    // Simulate diffusion & intercalation
    VtkOutput sim_output("elongation");
    for (auto time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        sim_output.write_positions(X, n_cells);
        sim_output.write_protrusions(prots, n_cells*LINKS_P_CELL);
        sim_output.write_type(cell_type, n_cells);
        // sim_output.write_polarity(X, n_cells);
        sim_output.write_field(X, n_cells, "Wnt");
        if (time_step == N_TIME_STEPS) return 0;

        // X.step(DELTA_T, d_potential, n_cells);
        X.step(DELTA_T, d_potential, intercalation, n_cells);
        proliferate<<<(n_cells + 128 - 1)/128, 128>>>(0.005, 0.733333);
        hipDeviceSynchronize();
        X.build_lattice(n_cells, R_LINK);
        update_links<<<(n_cells*LINKS_P_CELL + 32 - 1)/32, 32>>>(X.cell_id,
            X.cube_id, X.cube_start, X.cube_end);
        hipDeviceSynchronize();
    }
}
