#include "hip/hip_runtime.h"
// Simulate elongation of semisphere
#include <thread>
#include <functional>
#include <hiprand/hiprand_kernel.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/links.cuh"
#include "../lib/polarity.cuh"
#include "../lib/property.cuh"
#include "../lib/vtk.cuh"


const auto n_0 = 5000;
const auto n_max = 61000;
const auto r_max = 1;
const auto r_link = 1.5;
const auto links_per_cell = 1.f;  // Must be >= 1 as rand states used to proliferate
const auto link_strength = 0.5;
const auto n_time_steps = 500;
const auto skip_steps = 5;
const auto dt = 0.2;
enum Cell_types {mesenchyme, epithelium};

MAKE_PT(Lb_cell, x, y, z, w, phi, theta);


__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;  // number of mesenchymal neighbours
__device__ int* d_epi_nbs;

__device__ Lb_cell pairwise_interaction(Lb_cell Xi, Lb_cell Xj, int i, int j) {
    Lb_cell dF {0};
    if (i == j) {
        // D_ASSERT(Xi.w >= 0);
        dF.w = (d_type[i] > mesenchyme) - 0.01*Xi.w;
        return dF;
    }

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        F = fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2;
    } else {
        F = fmaxf(0.8 - dist, 0)*2 - fmaxf(dist - 0.9, 0)/2;
    }
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    auto D = dist < r_max ? 0.1 : 0;
    dF.w = - r.w*D;

    if (d_type[j] == mesenchyme) d_mes_nbs[i] += 1;
    else d_epi_nbs[i] += 1;

    if (d_type[i] == mesenchyme or d_type[j] == mesenchyme) return dF;

    dF += polarity_force(Xi, Xj)*0.2;
    return dF;
}

#include "../lib/solvers.cuh"


__global__ void update_links(const Lattice<n_max>* __restrict__ d_lattice,
        const Lb_cell* __restrict d_X, int n_cells, int n_links, Link* d_link,
        hiprandState* d_state) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= n_links) return;

    auto j = min(static_cast<int>(hiprand_uniform(&d_state[i])*n_cells),
        n_cells - 1);  // hiprand_uniform includes 1.0!
    auto rand_cube = d_lattice->d_cube_id[j]
        +  static_cast<int>(hiprand_uniform(&d_state[i])*3) - 1
        + (static_cast<int>(hiprand_uniform(&d_state[i])*3) - 1)*LATTICE_SIZE
        + (static_cast<int>(hiprand_uniform(&d_state[i])*3) - 1)*LATTICE_SIZE*LATTICE_SIZE;
    auto cells_in_cube = d_lattice->d_cube_end[rand_cube] - d_lattice->d_cube_start[rand_cube];
    if (cells_in_cube < 1) return;

    auto k = d_lattice->d_cube_start[rand_cube]
        + min(static_cast<int>(hiprand_uniform(&d_state[i])*cells_in_cube), cells_in_cube - 1);
    D_ASSERT(d_lattice->d_cell_id[j] >= 0); D_ASSERT(d_lattice->d_cell_id[j] < n_cells);
    D_ASSERT(d_lattice->d_cell_id[k] >= 0); D_ASSERT(d_lattice->d_cell_id[k] < n_cells);
    auto r = d_X[d_lattice->d_cell_id[j]] - d_X[d_lattice->d_cell_id[k]];
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if ((j != k) and (d_type[d_lattice->d_cell_id[j]] == mesenchyme)
            and (d_type[d_lattice->d_cell_id[k]] == mesenchyme)
            and (dist < r_link)
            and (fabs(r.w/(d_X[d_lattice->d_cell_id[j]].w + d_X[d_lattice->d_cell_id[k]].w)) > 0.2)) {
            // and (fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
        d_link[i].a = d_lattice->d_cell_id[j];
        d_link[i].b = d_lattice->d_cell_id[k];
    }
}


__global__ void proliferate(float rate, float mean_distance, Lb_cell* d_X,
        int* d_n_cells, hiprandState* d_state) {
    D_ASSERT(*d_n_cells*rate <= n_max);
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= *d_n_cells*(1 - rate)) return;  // Dividing new cells is problematic!

    switch (d_type[i]) {
        case mesenchyme: {
            auto r = hiprand_uniform(&d_state[i]);
            if (r > rate) return;
        }
        case epithelium: {
            if (d_epi_nbs[i] > d_mes_nbs[i]) return;
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto phi = hiprand_uniform(&d_state[i])*M_PI;
    auto theta = hiprand_uniform(&d_state[i])*2*M_PI;
    d_X[n].x = d_X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance/4*cosf(theta);
    d_X[n].w = d_X[i].w/2;
    d_X[i].w = d_X[i].w/2;
    d_X[n].phi = d_X[i].phi;
    d_X[n].theta = d_X[i].theta;
    d_type[n] = d_type[i];
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<Lb_cell, n_max, Lattice_solver> bolls(n_0);
    uniform_sphere(0.733333, bolls);
    Property<n_max, Cell_types> type;
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    for (auto i = 0; i < n_0; i++) {
        bolls.h_X[i].x = fabs(bolls.h_X[i].x);
        bolls.h_X[i].y = bolls.h_X[i].y/1.5;
        bolls.h_X[i].w = 0;
        type.h_prop[i] = mesenchyme;
    }
    bolls.copy_to_device();
    type.copy_to_device();
    Property<n_max, int> n_mes_nbs;
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    Property<n_max, int> n_epi_nbs;
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));
    Links<static_cast<int>(n_max*links_per_cell)> links(link_strength, n_0*links_per_cell);
    auto intercalation = std::bind(
        link_forces<static_cast<int>(n_max*links_per_cell), Lb_cell>,
        links, std::placeholders::_1, std::placeholders::_2);

    // Relax
    for (auto time_step = 0; time_step <= 200; time_step++) {
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_0, 0);
        bolls.take_step(dt);
    }

    // Find epithelium
    bolls.copy_to_host();
    n_mes_nbs.copy_to_host();
    for (auto i = 0; i < n_0; i++) {
        if (n_mes_nbs.h_prop[i] < 12*2 and bolls.h_X[i].x > 0) {  // 2nd order solver
            type.h_prop[i] = epithelium;
            auto dist = sqrtf(bolls.h_X[i].x*bolls.h_X[i].x
                + bolls.h_X[i].y*bolls.h_X[i].y + bolls.h_X[i].z*bolls.h_X[i].z);
            bolls.h_X[i].phi = atan2(bolls.h_X[i].y, bolls.h_X[i].x);
            bolls.h_X[i].theta = acosf(bolls.h_X[i].z/dist);
        } else {
            bolls.h_X[i].phi = 0;
            bolls.h_X[i].theta = 0;
        }
        bolls.h_X[i].w = 0;
    }
    bolls.copy_to_device();
    type.copy_to_device();
    bolls.take_step(dt);  // Relax epithelium before proliferate

    // Simulate diffusion & intercalation
    Vtk_output output("elongation");
    for (auto time_step = 0; time_step <= n_time_steps/skip_steps; time_step++) {
        bolls.copy_to_host();
        links.copy_to_host();
        type.copy_to_host();

        std::thread calculation([&] {
            for (auto i = 0; i < skip_steps; i++) {
                proliferate<<<(bolls.get_d_n() + 128 - 1)/128, 128>>>(0.005, 0.733333, bolls.d_X,
                    bolls.d_n, links.d_state);
                links.set_d_n(bolls.get_d_n()*links_per_cell);
                bolls.build_lattice(r_link);
                update_links<<<(links.get_d_n() + 32 - 1)/32, 32>>>(bolls.d_lattice,
                    bolls.d_X, bolls.get_d_n(), links.get_d_n(), links.d_link, links.d_state);
                thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + bolls.get_d_n(), 0);
                thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + bolls.get_d_n(), 0);
                bolls.take_step(dt, intercalation);
            }
        });

        output.write_positions(bolls);
        output.write_links(links);
        output.write_property(type);
        // output.write_polarity(bolls);
        output.write_field(bolls, "Wnt");

        calculation.join();
    }

    return 0;
}
