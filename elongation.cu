#include "hip/hip_runtime.h"
// Simulate elongation of semisphere
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/protrusions.cuh"
#include "../lib/epithelium.cuh"
#include "../lib/vtk.cuh"


const auto R_MAX = 1;
const auto R_MIN = 0.6;
const auto N_MAX = 61000;
const auto R_LINK = 1.5;
const auto LINKS_P_CELL = 1.f;
const auto N_TIME_STEPS = 500;
const auto DELTA_T = 0.2;
enum CELL_TYPES {MESENCHYME, STRETCHED_EPI, EPITHELIUM};

int n_cells;
__device__ auto d_n_cells = 5000;
__device__ __managed__ CELL_TYPES cell_type[N_MAX];


MAKE_DTYPE(lbcell, x, y, z, w, phi, theta);

Solution<lbcell, N_MAX, LatticeSolver> bolls;
Protrusions<static_cast<int>(N_MAX*LINKS_P_CELL)> links;


__device__ lbcell cubic_w_diffusion(lbcell Xi, lbcell Xj, int i, int j) {
    lbcell dF {0};
    if (i == j) {
        assert(Xi.w >= 0);
        dF.w = (cell_type[i] > MESENCHYME) - 0.01*Xi.w;
        return dF;
    }

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    auto F = 2*(R_MIN - dist)*(R_MAX - dist) + (R_MAX - dist)*(R_MAX - dist);
    dF.x = r.x*F/dist*(Xi.x > 0);
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    auto D = dist < R_MAX ? 0.1 : 0;
    dF.w = - r.w*D;

    if (cell_type[i] == MESENCHYME or cell_type[j] == MESENCHYME) return dF;

    if (dist < 0.733333) cell_type[i] = EPITHELIUM;
    dF += polarity_force(Xi, Xj)*0.2;
    return dF;
}

__device__ auto d_cubic_w_diffusion = &cubic_w_diffusion;
auto h_cubic_w_diffusion = get_device_object(d_cubic_w_diffusion, 0);


__device__ lbcell count_neighbours(lbcell Xi, lbcell Xj, int i, int j) {
    lbcell dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    dF.w = dist < R_MAX ? 1 : 0;
    return dF;
}

__device__ auto d_count_neighbours = &count_neighbours;
auto h_count_neighbours = get_device_object(d_count_neighbours, 0);


__global__ void update_links(const int* __restrict__ d_cell_id,
        const int* __restrict__ d_cube_id, const int* __restrict__ d_cube_start,
        const int* __restrict__ d_cube_end, const lbcell* __restrict d_X, Link* d_cell_ids,
        hiprandState* d_state) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= d_n_cells*LINKS_P_CELL) return;

    auto j = static_cast<int>(hiprand_uniform(&d_state[i])*d_n_cells);
    auto rand_cube = d_cube_id[j]
        +  static_cast<int>(hiprand_uniform(&d_state[i])*3) - 1
        + (static_cast<int>(hiprand_uniform(&d_state[i])*3) - 1)*LATTICE_SIZE
        + (static_cast<int>(hiprand_uniform(&d_state[i])*3) - 1)*LATTICE_SIZE*LATTICE_SIZE;
    auto cells_in_cube = d_cube_end[rand_cube] - d_cube_start[rand_cube];
    if (cells_in_cube < 1) return;

    auto k = d_cube_start[rand_cube]
        + static_cast<int>(hiprand_uniform(&d_state[i])*cells_in_cube);
    auto r = d_X[d_cell_id[j]] - d_X[d_cell_id[k]];
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if ((j != k) and (cell_type[d_cell_id[j]] == MESENCHYME)
            and (cell_type[d_cell_id[k]] == MESENCHYME)
            and (dist < R_LINK)
            and (fabs(r.w/(d_X[d_cell_id[j]].w + d_X[d_cell_id[k]].w)) > 0.2)) {
            // and (fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
        d_cell_ids[i].a = d_cell_id[j];
        d_cell_ids[i].b = d_cell_id[k];
    }
}

void intercalation(const lbcell* __restrict__ d_X, lbcell* d_dX) {
    link_force<<<(n_cells*LINKS_P_CELL + 32 - 1)/32, 32>>>(d_X, d_dX, links.d_cell_id,
        n_cells*LINKS_P_CELL, 0.5);
}


__global__ void proliferate(float rate, float mean_distance, lbcell* d_X, hiprandState* d_state) {
    assert(rate*d_n_cells <= N_MAX);
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= d_n_cells) return;

    if (cell_type[i] == EPITHELIUM) {
        cell_type[i] = STRETCHED_EPI;
        return;
    }

    if (cell_type[i] == MESENCHYME) {
        auto r = hiprand_uniform(&d_state[i]);
        if (r > rate) return;
    }

    auto n = atomicAdd(&d_n_cells, 1);
    auto phi = hiprand_uniform(&d_state[i])*M_PI;
    auto theta = hiprand_uniform(&d_state[i])*2*M_PI;
    d_X[n].x = d_X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance/4*cosf(theta);
    d_X[n].w = d_X[i].w/2;
    d_X[i].w = d_X[i].w/2;
    d_X[n].phi = d_X[i].phi;
    d_X[n].theta = d_X[i].theta;
    cell_type[n] = cell_type[i] == MESENCHYME ? MESENCHYME : STRETCHED_EPI;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    n_cells = get_device_object(d_n_cells);
    uniform_sphere(0.733333, bolls, n_cells);
    for (auto i = 0; i < n_cells; i++) {
        bolls.h_X[i].x = fabs(bolls.h_X[i].x);
        bolls.h_X[i].y = bolls.h_X[i].y/1.5;
        bolls.h_X[i].w = 0;
        cell_type[i] = MESENCHYME;
    }
    bolls.memcpyHostToDevice();

    // Relax
    VtkOutput relax_output("relaxation");
    for (auto time_step = 0; time_step <= 200; time_step++) {
        bolls.step(DELTA_T, h_cubic_w_diffusion, n_cells);
        relax_output.print_progress();
    }
    relax_output.print_done();

    // Find epithelium
    bolls.step(1, h_count_neighbours, n_cells);
    bolls.memcpyDeviceToHost();
    for (auto i = 0; i < n_cells; i++) {
        if (bolls.h_X[i].w < 12 and bolls.h_X[i].x > 0) {
            cell_type[i] = STRETCHED_EPI;
            auto dist = sqrtf(bolls.h_X[i].x*bolls.h_X[i].x
                + bolls.h_X[i].y*bolls.h_X[i].y + bolls.h_X[i].z*bolls.h_X[i].z);
            bolls.h_X[i].phi = atan2(bolls.h_X[i].y, bolls.h_X[i].x);
            bolls.h_X[i].theta = acosf(bolls.h_X[i].z/dist);
        } else {
            bolls.h_X[i].phi = 0;
            bolls.h_X[i].theta = 0;
        }
        bolls.h_X[i].w = 0;
    }
    bolls.memcpyHostToDevice();

    // Simulate diffusion & intercalation
    VtkOutput sim_output("elongation");
    for (auto time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        bolls.memcpyDeviceToHost();
        links.memcpyDeviceToHost();
        sim_output.write_positions(bolls, n_cells);
        sim_output.write_protrusions(links, n_cells*LINKS_P_CELL);
        sim_output.write_type(cell_type, n_cells);
        // sim_output.write_polarity(bolls, n_cells);
        sim_output.write_field(bolls, n_cells, "Wnt");

        bolls.step(DELTA_T, h_cubic_w_diffusion, intercalation, n_cells);
        proliferate<<<(n_cells + 128 - 1)/128, 128>>>(0.005, 0.733333, bolls.d_X, links.d_state);
        n_cells = get_device_object(d_n_cells);
        bolls.build_lattice(n_cells, R_LINK);
        update_links<<<(n_cells*LINKS_P_CELL + 32 - 1)/32, 32>>>(bolls.d_cell_id,
            bolls.d_cube_id, bolls.d_cube_start, bolls.d_cube_end, bolls.d_X,
            links.d_cell_id, links.d_state);
    }

    return 0;
}
