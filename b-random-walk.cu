#include "hip/hip_runtime.h"
// Simulate mono-polar migration
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/polarity.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"


const auto r_max = 1;
const auto n_cells = 500;
const auto n_time_steps = 150;
const auto dt = 0.05;


__device__ Po_cell relu_w_migration(
    Po_cell Xi, Po_cell r, float dist, int i, int j)
{
    Po_cell dF{0};
    if (i == j) return dF;

    if (dist > r_max) return dF;

    auto F = fmaxf(0.7 - dist, 0) * 2 - fmaxf(dist - 0.8, 0);
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    dF += migration_force(Xi, r, dist);
    return dF;
}


int main(int argc, const char* argv[])
{
    // Prepare initial state
    Solution<Po_cell, Tile_solver> cells{n_cells};
    relaxed_sphere(0.75, cells);
    cells.h_X[0].x = 0;
    cells.h_X[0].y = 0;
    cells.h_X[0].z = 0;
    cells.h_X[0].phi = 0.01;
    cells.copy_to_device();

    // Integrate cell positions
    Vtk_output output{"random-walk"};
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        cells.copy_to_host();
        cells.take_step<relu_w_migration>(dt);
        output.write_positions(cells);
        output.write_polarity(cells);
    }

    return 0;
}