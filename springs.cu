#include "hip/hip_runtime.h"
// Integrate N-body problem with springs between all bodies.
#include <assert.h>
#include <iostream>
#include <sstream>
#include <cmath>
#include <sys/stat.h>

#include "../lib/inits.cuh"
#include "../lib/vtk.cuh"
#include "../lib/n2n.cuh"


const float L_0 = 0.5; // Relaxed spring length
const float DELTA_T = 0.001;
const uint N_CELLS = 800;
const uint N_TIME_STEPS = 100;

__device__ __managed__ float3 X[N_CELLS], dX[N_CELLS], X1[N_CELLS], dX1[N_CELLS];


__device__ float3 neighbourhood_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 r;
    float3 dF = {0.0f, 0.0f, 0.0f};
    r.x = Xi.x - Xj.x;
    r.y = Xi.y - Xj.y;
    r.z = Xi.z - Xj.z;
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (i != j) {
        dF.x = r.x*(L_0 - dist)/dist;
        dF.y = r.y*(L_0 - dist)/dist;
        dF.z = r.z*(L_0 - dist)/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


void global_interactions(const __restrict__ float3* X, float3* dX) {}


int main(int argc, const char* argv[]) {
    assert(N_CELLS % TILE_SIZE == 0);

    // Prepare initial state
    uniform_sphere(N_CELLS, L_0, X);

    // Integrate positions
    VtkOutput output("springs");
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(N_CELLS, X);

        if (time_step < N_TIME_STEPS) {
            heun_step(DELTA_T, N_CELLS, X, dX, X1, dX1);
        }
    }

    return 0;
}
