#include "hip/hip_runtime.h"
// Integrate N-body problem with springs between all bodies.
#include <assert.h>
#include <iostream>
#include <sstream>
#include <cmath>
#include <sys/stat.h>

#include "../lib/vtk.cuh"
#include "../lib/n2n.cuh"


const float L_0 = 0.5; // Relaxed spring length
const float delta_t = 0.001;
const uint N_CELLS = 800;
const uint N_TIME_STEPS = 100;

__device__ __managed__ float3 X[N_CELLS];


__device__ float3 cell_cell_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 r;
    float3 dF = {0.0f, 0.0f, 0.0f};
    r.x = Xi.x - Xj.x;
    r.y = Xi.y - Xj.y;
    r.z = Xi.z - Xj.z;
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > 1e-7) {
        dF.x += r.x*(L_0 - dist)/dist;
        dF.y += r.y*(L_0 - dist)/dist;
        dF.z += r.z*(L_0 - dist)/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


int main(int argc, const char* argv[]) {
    assert(N_CELLS % TILE_SIZE == 0);

    // Prepare initial state
    float r_max = pow(N_CELLS/0.75, 1./3)*L_0/2; // Sphere packing
    for (int i = 0; i < N_CELLS; i++) {
        float r = r_max*pow(rand()/(RAND_MAX + 1.), 1./3);
        float theta = rand()/(RAND_MAX + 1.)*2*M_PI;
        float phi = acos(2.*rand()/(RAND_MAX + 1.) - 1);
        X[i].x = r*sin(theta)*sin(phi);
        X[i].y = r*cos(theta)*sin(phi);
        X[i].z = r*cos(phi);
    }

    // Integrate positions
    mkdir("output", 755);
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        char file_name[22];
        sprintf(file_name, "output/springs_%03i.vtk", time_step);
        write_positions(file_name, N_CELLS, X);

        if (time_step < N_TIME_STEPS) {
            euler_step(delta_t, N_CELLS, X);
        }
    }

    return 0;
}
