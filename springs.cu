#include "hip/hip_runtime.h"
// Integrate N-body problem with springs between all bodies. Parallelization
// after http://http.developer.nvidia.com/GPUGems3/gpugems3_ch31.html.

#include <assert.h>
#include <iostream>
#include <sstream>
#include <cmath>
#include <sys/stat.h>

#include "../lib/vtk.cu"

#define N_CELLS 800
#define TILE_SIZE 16
#define N_TIME_STEPS 100


__device__ __managed__ float3 X[N_CELLS];

__device__ float3 body_body_force(float3 Xi, float3 Xj) {
    float3 r;
    float3 dF = {0.0f, 0.0f, 0.0f};
    r.x = Xj.x - Xi.x;
    r.y = Xj.y - Xi.y;
    r.z = Xj.z - Xi.z;
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > 1e-8) {
        dF.x += r.x*(dist - 0.5)/dist;
        dF.y += r.y*(dist - 0.5)/dist;
        dF.z += r.z*(dist - 0.5)/dist;
    }
    return dF;
}

// Calculate new X one thread per cell, to TILE_SIZE other cells at a time
__global__ void integrate_step() {
    __shared__ float3 shX[TILE_SIZE];
    int cell_idx = blockIdx.x*blockDim.x + threadIdx.x;
    float3 Xi = X[cell_idx];
    float3 Fi = {0.0f, 0.0f, 0.0f};
    for (int tile_start = 0; tile_start < N_CELLS; tile_start += TILE_SIZE) {
        int other_cell_idx = tile_start + threadIdx.x;
        shX[threadIdx.x] = X[other_cell_idx];
        __syncthreads();
        for (int i = 0; i < TILE_SIZE; i++) {
            float3 dF = body_body_force(Xi, shX[i]);
            Fi.x += dF.x;
            Fi.y += dF.y;
            Fi.z += dF.z;
        }
    }
    X[cell_idx].x = Xi.x + Fi.x*0.001;
    X[cell_idx].y = Xi.y + Fi.y*0.001;
    X[cell_idx].z = Xi.z + Fi.z*0.001;
}

int main(int argc, const char* argv[]) {
    assert(N_CELLS % TILE_SIZE == 0);

    // Prepare initial state
    float r_min = 0.5;
    float r_max = pow(N_CELLS/0.75, 1./3)*r_min/2; // Sphere packing
    for (int i = 0; i < N_CELLS; i++) {
        float r = r_max*rand()/(RAND_MAX + 1.);
        float theta = rand()/(RAND_MAX + 1.)*2*M_PI;
        float phi = acos(2.*rand()/(RAND_MAX + 1.) - 1);
        X[i].x = r*sin(theta)*sin(phi);
        X[i].y = r*cos(theta)*sin(phi);
        X[i].z = r*cos(phi);
    }

    // Integrate cell positions
    mkdir("output", 755);
    int n_blocks = (N_CELLS + TILE_SIZE - 1)/TILE_SIZE; // ceil int div.
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        std::stringstream file_name;
        file_name << "output/springs_" << time_step << ".vtk";
        write_positions(file_name.str().c_str(), N_CELLS, X);

        if (time_step < N_TIME_STEPS) {
            integrate_step<<<n_blocks, TILE_SIZE>>>();
            hipDeviceSynchronize();
        }
    }

    return 0;
}
