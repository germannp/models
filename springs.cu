#include "hip/hip_runtime.h"
// Integrate N-body problem with springs between all bodies
#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"


const auto L_0 = 0.5f;  // Relaxed spring length
const auto N_CELLS = 800u;
const auto N_TIME_STEPS = 100u;
const auto DELTA_T = 0.001f;


__device__ float3 spring(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    dF = r*(L_0 - dist)/dist;
    return dF;
}

__device__ auto d_spring = &spring;
auto h_spring = get_device_object(d_spring, 0);


int main(int argc, const char* argv[]) {
    // Prepare initial state
    Solution<float3, N_CELLS, N2nSolver> bolls;
    uniform_sphere(L_0, bolls);

    // Integrate positions
    VtkOutput output("springs");
    for (auto time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        bolls.memcpyDeviceToHost();
        bolls.step(DELTA_T, h_spring);  // Writing starts during calculation, use thread
        output.write_positions(bolls);  // for full concurency.
    }

    return 0;
}
