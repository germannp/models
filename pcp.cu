#include "hip/hip_runtime.h"
// Simulate planer cell polarity aligned by gradient
#include "../lib/dtypes.cuh"
#include "../lib/solvers.cuh"
#include "../lib/inits.cuh"
#include "../lib/vtk.cuh"
#include "../lib/polarity.cuh"


const auto r_max = 1;
const auto r_min = 0.6;
const auto D = 0.5f;
const auto n_cells = 500;
const auto n_time_steps = 300;
const auto dt = 0.025;

MAKE_PT(Po_cell4, x, y, z, w, theta, phi);


__device__ Po_cell4 biased_pcp(Po_cell4 Xi, Po_cell4 Xj, int i, int j) {
    Po_cell4 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = norm3df(r.x, r.y, r.z);
    if (dist > r_max) return dF;

    auto F = 2*(r_min - dist)*(r_max - dist) + powf(r_max - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    dF.w = i == 0 ? 0 : -r.w*D;

    // U_PCP = - Σ(n_i . n_j)^2/2
    add_pcp_force(Xi, Xj, dF);
    if (r.w > 0) return dF;

    // U_WNT = - ΣXj.w*(n_i . r_ij/r)^2/2 to bias along w
    Polarity rhat {acosf(-r.z/dist), atan2(-r.y, -r.x)};
    add_pcp_force(Xi, rhat, dF, Xj.w);
    return dF;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<Po_cell4, n_cells, Lattice_solver> bolls;
    for (auto i = 0; i < n_cells; i++) {
        bolls.h_X[i].w = (i == 0)*10;
        bolls.h_X[i].theta = acos(2.*rand()/(RAND_MAX + 1.) - 1.);
        bolls.h_X[i].phi = 2.*M_PI*rand()/(RAND_MAX + 1.);
    }
    uniform_sphere(0.5, bolls);

    // Integrate cell positions
    Vtk_output output("pcp");
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        bolls.copy_to_host();
        bolls.take_step<biased_pcp>(dt);
        output.write_positions(bolls);
        output.write_field(bolls);
        output.write_polarity(bolls);
    }

    return 0;
}
