#include "hip/hip_runtime.h"
// Simulating a layer.
#include <assert.h>
#include <cmath>
#include <sys/stat.h>
#include <iostream>

#include "../lib/inits.cuh"
#include "../lib/vtk.cuh"
// #include "../lib/n2n.cuh"
#include "../lib/lattice.cuh"


const float R_MAX = 1;
const float R_MIN = 0.6;
const int N_CELLS = 1000;
const int N_TIME_STEPS = 200;
const float DELTA_T = 0.005;

__device__ __managed__ float3 X[N_CELLS], dX[N_CELLS], X1[N_CELLS], dX1[N_CELLS];


__device__ float3 neighbourhood_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF = {0.0f, 0.0f, 0.0f};
    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = fminf(sqrtf(r.x*r.x + r.y*r.y + r.z*r.z), R_MAX);
    if (i != j) {
        int n = 2;
        float strength = 100;
        float F = strength*n*(R_MIN - dist)*powf(R_MAX - dist, n - 1)
            + strength*powf(R_MAX - dist, n);
        dF.x = r.x*F/dist;
        dF.y = r.y*F/dist;
        dF.z = r.z*F/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


void global_interactions(const __restrict__ float3* X, float3* dX) {}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_circle(N_CELLS, 0.733333/1.5, X);
    for (int i = 0; i < N_CELLS; i++) {
        X[i].x = sin(X[i].y);
    }

    // Integrate cell positions
    VtkOutput output("layer");
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(N_CELLS, X);

        if (time_step*DELTA_T <= 1) {
            heun_step(DELTA_T, N_CELLS, X, dX, X1, dX1);
        }
    }

    return 0;
}
