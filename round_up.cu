#include "hip/hip_runtime.h"
// Simulating rounding up.
#include <assert.h>
#include <cmath>
#include <sys/stat.h>

#include "../lib/inits.cuh"
#include "../lib/vtk.cuh"
// #include "../lib/n2n.cuh"
#include "../lib/lattice.cuh"


const float R_MAX = 1;
const float R_MIN = 0.6;
const int N_CELLS = 100;
const float DELTA_T = 0.005;

__device__ __managed__ float3 X[N_CELLS], dX[N_CELLS], X1[N_CELLS], dX1[N_CELLS], X1[N_CELLS], dX1[N_CELLS];
__device__ __managed__ int time_step;


__device__ float3 neighbourhood_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF = {0.0f, 0.0f, 0.0f};
    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = fminf(sqrtf(r.x*r.x + r.y*r.y + r.z*r.z), R_MAX);
    if (i != j) {
        int n = 2;
        float strength = 100;
        float F = strength*n*(R_MIN - dist)*powf(R_MAX - dist, n - 1)
            + strength*powf(R_MAX - dist, n);
        dF.x = r.x*F/dist;
        dF.y = r.y*F/dist;
        dF.z = r.z*F/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


// Smooth transition from step(x < 0) = 0 to step(x > 0) = 1 over dx
__device__ float step(float x) {
    float dx = 0.1;
    x = __saturatef((x + dx/2)/dx);
    return x*x*(3 - 2*x);
}

__global__ void squeeze_to_floor(const __restrict__ float3* X, float3* dX) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    float time = time_step*DELTA_T;
    if (i < N_CELLS) {
        dX[i].z += 10*step(-2 - X[i].z); // Floor
        if ((time >= 0.1) && (time <= 0.5)) {
            dX[i].z -= 10*step(X[i].z - (2 - (time - 0.1)/0.3));
        }
    }
}

void global_interactions(const __restrict__ float3* X, float3* dX) {
    squeeze_to_floor<<<(N_CELLS + 16 - 1)/16, 16>>>(X, dX);
    hipDeviceSynchronize();
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_circle(N_CELLS, 0.733333, X);
    // uniform_sphere(N_CELLS, 0.733333, X);

    // Integrate cell positions
    VtkOutput output("round_up");
    for (time_step = 0; time_step*DELTA_T <= 1; time_step++) {
        output.write_positions(N_CELLS, X);

        if (time_step*DELTA_T <= 1) {
            heun_step(DELTA_T, N_CELLS, X, dX, X1, dX1);
        }
    }

    return 0;
}
