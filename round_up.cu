#include "hip/hip_runtime.h"
// Simulate rounding up
#include <assert.h>
#include <cmath>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"


const float R_MAX = 1;
const float R_MIN = 0.6;
const int N_CELLS = 100;
const float DELTA_T = 0.005;

__device__ __managed__ Solution<float3, N_CELLS, LatticeSolver> X;
__device__ __managed__ int time_step;


__device__ float3 clipped_polynomial(float3 Xi, float3 Xj, int i, int j) {
    float3 dF = {0.0f, 0.0f, 0.0f};
    if (i == j) return dF;

    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    int n = 2;
    float strength = 100;
    float F = strength*n*(R_MIN - dist)*powf(R_MAX - dist, n - 1)
        + strength*powf(R_MAX - dist, n);
    // float F = strength*(fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    assert(dF.x == dF.x);  // For NaN f != f.
    return dF;
}

__device__ __managed__ nhoodint<float3> potential = clipped_polynomial;


// Smooth transition from step(x < 0) = 0 to step(x > 0) = 1 over dx
__device__ float step(float x) {
    float dx = 0.1;
    x = __saturatef((x + dx/2)/dx);
    return x*x*(3 - 2*x);
}

__global__ void squeeze_kernel(const float3* __restrict__ X, float3* dX) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N_CELLS) return;

    float time = time_step*DELTA_T;
    dX[i].z += 10*step(-2 - X[i].z);  // Floor
    if ((time >= 0.1) and (time <= 0.5)) {
        dX[i].z -= 10*step(X[i].z - (2 - (time - 0.1)/0.3));
    }
}

void squeeze_to_floor(const float3* __restrict__ X, float3* dX) {
    squeeze_kernel<<<(N_CELLS + 16 - 1)/16, 16>>>(X, dX);
    hipDeviceSynchronize();
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_circle(0.733333, X);
    // uniform_sphere(0.733333, X);

    // Integrate cell positions
    VtkOutput output("round_up");
    for (time_step = 0; time_step*DELTA_T <= 1; time_step++) {
        output.write_positions(X);
        if (time_step*DELTA_T == 1) return 0;

        X.step(DELTA_T, potential, squeeze_to_floor);
    }
}
