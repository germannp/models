#include "hip/hip_runtime.h"
// Simulate rounding up
#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/vtk.cuh"


const auto R_MAX = 1;
const auto R_MIN = 0.6;
const auto N_CELLS = 100;
const auto DELTA_T = 0.005;
auto time_step = 0;


__device__ float3 pairwise_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    auto n = 2;
    auto strength = 100;
    auto F = strength*n*(R_MIN - dist)*powf(R_MAX - dist, n - 1)
        + strength*powf(R_MAX - dist, n);
    // auto F = strength*(fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2);
    dF = r*F/dist;
    return dF;
}

#include "../lib/solvers.cuh"


// Smooth transition from step(x < 0) = 0 to step(x > 0) = 1 over dx
__device__ float step(float x) {
    auto dx = 0.1;
    x = __saturatef((x + dx/2)/dx);
    return x*x*(3 - 2*x);
}

__global__ void squeeze_kernel(const float3* __restrict__ bolls, float3* dX,
        int time_step) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N_CELLS) return;

    auto time = time_step*DELTA_T;
    dX[i].z += 10*step(-2 - bolls[i].z);  // Floor
    if ((time >= 0.1) and (time <= 0.5)) {
        dX[i].z -= 10*step(bolls[i].z - (2 - (time - 0.1)/0.3));
    }
}

void squeeze_to_floor(const float3* __restrict__ d_X, float3* d_dX) {
    squeeze_kernel<<<(N_CELLS + 16 - 1)/16, 16>>>(d_X, d_dX, time_step);
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<float3, N_CELLS, LatticeSolver> bolls;
    uniform_circle(0.733333, bolls);
    // uniform_sphere(0.733333, bolls);

    // Integrate cell positions
    VtkOutput output("round_up");
    for (time_step = 0; time_step*DELTA_T <= 1; time_step++) {
        bolls.memcpyDeviceToHost();
        bolls.step(DELTA_T, squeeze_to_floor);
        output.write_positions(bolls);
    }

    return 0;
}
