#include "hip/hip_runtime.h"
// Simulate growing mesenchyme envelopped by epithelium
#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"
#include "../lib/epithelium.cuh"


const float R_MAX = 1;
const float RATE = 0.006;
const float MEAN_DIST = 0.75;
const int N_MAX = 5000;
const int N_TIME_STEPS = 500;
const float DELTA_T = 0.2;
enum CELL_TYPES {MESENCHYME, EPITHELIUM};

__device__ __managed__ Solution<pocell, N_MAX, LatticeSolver> X;
__device__ __managed__ CELL_TYPES cell_type[N_MAX];
__device__ __managed__ int n_neighbrs[N_MAX];
__device__ __managed__ int n_cells = 200;
__device__ hiprandState rand_states[N_MAX];


__device__ pocell cubic_w_polarity(pocell Xi, pocell Xj, int i, int j) {
    pocell dF = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    if (i == j) return dF;

    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    float F;
    if (cell_type[i] == cell_type[j]) {
        F = fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2;
    } else {
        F = fmaxf(0.8 - dist, 0)*2 - fmaxf(dist - 0.9, 0)/2;
    }
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    assert(dF.x == dF.x);  // For NaN f != f.

    if (cell_type[i] == MESENCHYME) {
        n_neighbrs[i] += 1;
        return dF;
    }
    if (cell_type[j] == MESENCHYME) return dF;

    n_neighbrs[i] += 1;  // Count only EPITHELIUM neighbours for EPITHELIUM
    dF = dF + polarity_force(Xi, Xj)*0.2;
    return dF;
}

__device__ __managed__ nhoodint<pocell> p_potential = cubic_w_polarity;


__global__ void reset_n_neighbrs() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n_cells) n_neighbrs[i] = 0;
}

__global__ void setup_rand_states() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N_MAX) hiprand_init(1337, i, 0, &rand_states[i]);
}

__global__ void proliferate(float rate, float mean_distance) {
    assert(rate*n_cells <= N_MAX);
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= n_cells) return;

    switch (cell_type[i]) {
        case MESENCHYME: {
            float r = hiprand_uniform(&rand_states[i]);
            if (r > rate) return;
            break;
        }
        case EPITHELIUM: {
            if (n_neighbrs[i] > 4*2) return;  // 2nd order solver
        }
    }

    int n = atomicAdd(&n_cells, 1);
    float phi = hiprand_uniform(&rand_states[i])*M_PI;
    float theta = hiprand_uniform(&rand_states[i])*2*M_PI;
    X[n].x = X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    X[n].y = X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    X[n].z = X[i].z + mean_distance/4*cosf(theta);
    X[n].phi = X[i].phi;
    X[n].theta = X[i].theta;
    cell_type[n] = cell_type[i];
    n_neighbrs[n] = 0;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(MEAN_DIST, X, n_cells);
    for (int i = 0; i < n_cells; i++) {
        cell_type[i] = MESENCHYME;
        n_neighbrs[i] = 0;
    }
    setup_rand_states<<<(N_MAX + 128 - 1)/128, 128>>>();
    hipDeviceSynchronize();

    // Relax
    for (int time_step = 0; time_step <= 500; time_step++) {
        reset_n_neighbrs<<<(n_cells + 128 - 1)/128, 128>>>();
        hipDeviceSynchronize();
        X.step(DELTA_T, p_potential, n_cells);
    }

    // Find epithelium
    for (int i = 0; i < n_cells; i++) {
        if (n_neighbrs[i] < 11*2) {  // 2nd order solver
            cell_type[i] = EPITHELIUM;
            float dist = sqrtf(X[i].x*X[i].x + X[i].y*X[i].y + X[i].z*X[i].z);
            X[i].phi = atan2(X[i].y, X[i].x);
            X[i].theta = acosf(X[i].z/dist);
        } else {
            X[i].phi = 0;
            X[i].theta = 0;
        }
    }

    // Simulate growth
    VtkOutput sim_output("passive_growth");
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        sim_output.write_positions(X, n_cells);
        sim_output.write_type(cell_type, n_cells);
        sim_output.write_polarity(X, n_cells);
        if (time_step == N_TIME_STEPS) return 0;

        reset_n_neighbrs<<<(n_cells + 128 - 1)/128, 128>>>();
        hipDeviceSynchronize();
        X.step(DELTA_T, p_potential, n_cells);
        proliferate<<<(n_cells + 128 - 1)/128, 128>>>(RATE, MEAN_DIST);
        hipDeviceSynchronize();
    }
}
