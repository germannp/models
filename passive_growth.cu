#include "hip/hip_runtime.h"
// Simulate growing mesenchyme envelopped by epithelium
#include <hiprand/hiprand_kernel.h>
#include <thrust/fill.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/property.cuh"
#include "../lib/protrusions.cuh"
#include "../lib/vtk.cuh"
#include "../lib/epithelium.cuh"


const auto R_MAX = 1;
const auto MEAN_DIST = 0.75;
const auto RATE = 0.006;
const auto N_MAX = 5000;
const auto N_TIME_STEPS = 500;
const auto DELTA_T = 0.2;
enum CELL_TYPES {MESENCHYME, EPITHELIUM};

Solution<pocell, N_MAX, LatticeSolver> bolls;
Property<N_MAX, CELL_TYPES> type;
Property<N_MAX, int> n_mes_nbs;
Property<N_MAX, int> n_epi_nbs;
hiprandState *d_state;


__device__ CELL_TYPES* d_type;
__device__ int* d_mes_nbs;
__device__ int* d_epi_nbs;

__device__ pocell relu_w_polarity(pocell Xi, pocell Xj, int i, int j) {
    pocell dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        F = fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2;
    } else {
        F = fmaxf(0.8 - dist, 0)*2 - fmaxf(dist - 0.9, 0)/2;
    }
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    if (d_type[j] == MESENCHYME) d_mes_nbs[i] += 1;
    else d_epi_nbs[i] += 1;

    if (d_type[i] == MESENCHYME or d_type[j] == MESENCHYME) return dF;

    dF += polarity_force(Xi, Xj)*0.2;
    return dF;
}

__device__ auto d_relu_w_polarity = &relu_w_polarity;
auto h_relu_w_polarity = get_device_object(d_relu_w_polarity, 0);


__global__ void proliferate(float rate, float mean_distance, pocell* d_X, int* d_n_cells,
        hiprandState* d_state) {
    D_ASSERT(*d_n_cells*rate <= N_MAX);
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= *d_n_cells*(1 - rate)) return;  // Dividing new cells is problematic!

    switch (d_type[i]) {
        case MESENCHYME: {
            auto r = hiprand_uniform(&d_state[i]);
            if (r > rate) return;
        }
        case EPITHELIUM: {
            if (d_epi_nbs[i] > d_mes_nbs[i]) return;
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto phi = hiprand_uniform(&d_state[i])*M_PI;
    auto theta = hiprand_uniform(&d_state[i])*2*M_PI;
    d_X[n].x = d_X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance/4*cosf(theta);
    d_X[n].phi = d_X[i].phi;
    d_X[n].theta = d_X[i].theta;
    d_type[n] = d_type[i] == MESENCHYME ? MESENCHYME : EPITHELIUM;
    d_mes_nbs[n] = 0;
    d_epi_nbs[n] = 0;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    bolls.set_n(200);
    uniform_sphere(MEAN_DIST, bolls);
    for (auto i = 0; i < bolls.get_n(); i++) type.h_prop[i] = MESENCHYME;
    type.memcpyHostToDevice();
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));
    hipMalloc(&d_state, N_MAX*sizeof(hiprandState));
    setup_rand_states<<<(N_MAX + 128 - 1)/128, 128>>>(d_state, N_MAX);

    // Relax
    for (auto time_step = 0; time_step <= 500; time_step++) {
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + bolls.get_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + bolls.get_n(), 0);
        bolls.step(DELTA_T, h_relu_w_polarity);
    }

    // Find epithelium
    bolls.memcpyDeviceToHost();
    n_mes_nbs.memcpyDeviceToHost();
    for (auto i = 0; i < bolls.get_n(); i++) {
        if (n_mes_nbs.h_prop[i] < 12*2) {  // 2nd order solver
            type.h_prop[i] = EPITHELIUM;
            auto dist = sqrtf(bolls.h_X[i].x*bolls.h_X[i].x + bolls.h_X[i].y*bolls.h_X[i].y
                + bolls.h_X[i].z*bolls.h_X[i].z);
            bolls.h_X[i].phi = atan2(bolls.h_X[i].y, bolls.h_X[i].x);
            bolls.h_X[i].theta = acosf(bolls.h_X[i].z/dist);
        } else {
            bolls.h_X[i].phi = 0;
            bolls.h_X[i].theta = 0;
        }
    }
    bolls.memcpyHostToDevice();
    type.memcpyHostToDevice();

    // Simulate growth
    VtkOutput sim_output("passive_growth");
    for (auto time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        bolls.memcpyDeviceToHost();
        type.memcpyDeviceToHost();
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + bolls.get_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + bolls.get_n(), 0);
        bolls.step(DELTA_T, h_relu_w_polarity);
        proliferate<<<(bolls.get_n() + 128 - 1)/128, 128>>>(RATE*(time_step > 100),
            MEAN_DIST, bolls.d_X, bolls.d_n, d_state);
        sim_output.write_positions(bolls);
        sim_output.write_property(type);
        sim_output.write_polarity(bolls);
    }

    return 0;
}
