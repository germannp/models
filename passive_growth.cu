#include "hip/hip_runtime.h"
// Simulate growing mesenchyme envelopped by epithelium
#include <hiprand/hiprand_kernel.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "../lib/dtypes.cuh"
#include "../lib/solvers.cuh"
#include "../lib/inits.cuh"
#include "../lib/property.cuh"
#include "../lib/links.cuh"
#include "../lib/vtk.cuh"
#include "../lib/polarity.cuh"


const auto r_max = 1;
const auto mean_dist = 0.75;
const auto prolif_rate = 0.006;
const auto n_0 = 200;
const auto n_max = 5000;
const auto n_time_steps = 500;
const auto dt = 0.2;
enum Cell_types {mesenchyme, epithelium};


__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;  // number of mesenchymal neighbours
__device__ int* d_epi_nbs;

__device__ Po_cell relu_w_epithelium(Po_cell Xi, Po_cell Xj, int i, int j) {
    Po_cell dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = norm3df(r.x, r.y, r.z);
    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        F = fmaxf(0.7 - dist, 0)*2 - fmaxf(dist - 0.8, 0)/2;
    } else {
        F = fmaxf(0.8 - dist, 0)*2 - fmaxf(dist - 0.9, 0)/2;
    }
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    if (d_type[j] == mesenchyme) d_mes_nbs[i] += 1;
    else d_epi_nbs[i] += 1;

    if (d_type[i] == mesenchyme or d_type[j] == mesenchyme) return dF;

    dF += rigidity_force(Xi, Xj)*0.2;
    return dF;
}


__global__ void proliferate(float rate, float mean_distance, Po_cell* d_X, int* d_n_cells,
        hiprandState* d_state) {
    D_ASSERT(*d_n_cells*rate <= n_max);
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= *d_n_cells*(1 - rate)) return;  // Dividing new cells is problematic!

    switch (d_type[i]) {
        case mesenchyme: {
            auto r = hiprand_uniform(&d_state[i]);
            if (r > rate) return;
        }
        case epithelium: {
            if (d_epi_nbs[i] > d_mes_nbs[i]) return;
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto phi = hiprand_uniform(&d_state[i])*M_PI;
    auto theta = hiprand_uniform(&d_state[i])*2*M_PI;
    d_X[n].x = d_X[i].x + mean_distance/4*sinf(theta)*cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance/4*sinf(theta)*sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance/4*cosf(theta);
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
    d_mes_nbs[n] = 0;
    d_epi_nbs[n] = 0;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<Po_cell, n_max, Lattice_solver> bolls(n_0);
    uniform_sphere(mean_dist, bolls);
    Property<n_max, Cell_types> type;
    for (auto i = 0; i < n_0; i++) type.h_prop[i] = mesenchyme;
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    type.copy_to_device();
    Property<n_max, int> n_mes_nbs;
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    Property<n_max, int> n_epi_nbs;
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));
    hiprandState *d_state;
    hipMalloc(&d_state, n_max*sizeof(hiprandState));
    setup_rand_states<<<(n_max + 128 - 1)/128, 128>>>(d_state, n_max);

    // Relax
    for (auto time_step = 0; time_step <= 500; time_step++) {
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_0, 0);
        bolls.take_step<relu_w_epithelium>(dt);
    }

    // Find epithelium
    bolls.copy_to_host();
    n_mes_nbs.copy_to_host();
    for (auto i = 0; i < n_0; i++) {
        if (n_mes_nbs.h_prop[i] < 12*2) {  // 2nd order solver
            type.h_prop[i] = epithelium;
            auto dist = sqrtf(bolls.h_X[i].x*bolls.h_X[i].x + bolls.h_X[i].y*bolls.h_X[i].y
                + bolls.h_X[i].z*bolls.h_X[i].z);
            bolls.h_X[i].theta = acosf(bolls.h_X[i].z/dist);
            bolls.h_X[i].phi = atan2(bolls.h_X[i].y, bolls.h_X[i].x);
        } else {
            bolls.h_X[i].theta = 0;
            bolls.h_X[i].phi = 0;
        }
    }
    bolls.copy_to_device();
    type.copy_to_device();

    // Simulate growth
    Vtk_output sim_output("passive_growth");
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        bolls.copy_to_host();
        type.copy_to_host();
        thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + bolls.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + bolls.get_d_n(), 0);
        bolls.take_step<relu_w_epithelium>(dt);
        proliferate<<<(bolls.get_d_n() + 128 - 1)/128, 128>>>(prolif_rate*(time_step > 100),
            mean_dist, bolls.d_X, bolls.d_n, d_state);
        sim_output.write_positions(bolls);
        sim_output.write_property(type);
        sim_output.write_polarity(bolls);
    }

    return 0;
}
