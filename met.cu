#include "hip/hip_runtime.h"
// Simulate a mesenchyme-to-epithelium transition
#include <assert.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"
#include "../lib/epithelium.cuh"


const float R_MAX = 1;
const float R_MIN = 0.6;
const int N_CELLS = 250;
const int N_TIME_STEPS = 100;
const float DELTA_T = 0.1;

__device__ __managed__ Solution<pocell, N_CELLS, LatticeSolver> X;


// Cubic potential plus k*(n_i . r_ij/r)^2/2 for all r_ij <= R_MAX
__device__ pocell epithelium(pocell Xi, pocell Xj, int i, int j) {
    pocell dF = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    if (i == j) return dF;

    pocell r = Xi - Xj;
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    float F = 2*(R_MIN - dist)*(R_MAX - dist) + powf(R_MAX - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;
    assert(dF.x == dF.x);  // For NaN f != f.

    dF += polarity_force(Xi, Xj)*0.2;
    return dF;
}

__device__ __managed__ nhoodint<pocell> potential = epithelium;


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(0.733333, X);
    for (int i = 0; i < N_CELLS; i++) {
        float dist = sqrtf(X[i].x*X[i].x + X[i].y*X[i].y + X[i].z*X[i].z);
        X[i].phi = atan2(X[i].y, X[i].x) + rand()/(RAND_MAX + 1.)*0.5;
        X[i].theta = acosf(X[i].z/dist) + rand()/(RAND_MAX + 1.)*0.5;
    }

    // Integrate cell positions
    VtkOutput output("epithelium");
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(X);
        output.write_polarity(X);
        if (time_step == N_TIME_STEPS) return 0;

        X.step(DELTA_T, potential);
    }
}
