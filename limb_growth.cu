#include "hip/hip_runtime.h"
// Simulation of limb bud growth starting with realistic limb bud shape

// Command line arguments
// argv[1]=input file tag
// argv[2]=output file tag
// argv[3]=proliferation rate
// argv[4]=time steps
// argv[4]=proliferation rate distr. (0=uniform, 1=PD gradient)

#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <iostream>
#include <list>
#include <sstream>
#include <string>
#include <vector>

#include "../../include/dtypes.cuh"
#include "../../include/inits.cuh"
#include "../../include/links.cuh"
#include "../../include/polarity.cuh"
#include "../../include/property.cuh"
#include "../../include/solvers.cuh"
#include "../../include/vtk.cuh"

const auto r_max = 1.0;
const auto r_min = 0.8;
const auto dt = 0.1f;
const auto n_max = 150000;
const auto prots_per_cell = 1;
const auto protrusion_strength = 0.2f;
const auto r_protrusion = 2.0f;

enum Cell_types { mesenchyme, epithelium, aer };

__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;
__device__ int* d_epi_nbs;
__device__ float* d_prolif_rate;

Property<n_max, int> n_mes_nbs("n_mes_nbs");  // defining these here so function
Property<n_max, int> n_epi_nbs("n_epi_nbs");  // "neighbour_init" can see them

MAKE_PT(Cell, w, f, theta, phi);

__device__ Cell wall_force(Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    if (i == j) {
        // D_ASSERT(Xi.w >= 0);
        dF.w = -0.01 * (d_type[i] == mesenchyme) * Xi.w;
        dF.f = -0.01 * (d_type[i] == mesenchyme) * Xi.f;
        if (Xi.w < 0.f) dF.w = 0.f;
        if (Xi.f < 0.f) dF.f = 0.f;
        return dF;
    }

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        if (d_type[i] == mesenchyme)
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0);
        else
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0) * 2.f;
    } else if (d_type[i] > mesenchyme && d_type[j] > mesenchyme) {
        F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0) * 2.f;
    } else {
        F = fmaxf(0.9 - dist, 0) * 2.f - fmaxf(dist - 0.9, 0) * 3.f;
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    dF.w = -r.w * (d_type[i] == mesenchyme) * 0.5f;
    dF.f = -r.f * (d_type[i] == mesenchyme) * 0.5f;

    if (d_type[i] >= epithelium && d_type[j] >= epithelium)
        dF += rigidity_force(Xi, r, dist) * 0.1f;


    if (d_type[j] >= epithelium)
        atomicAdd(&d_epi_nbs[i], 1);
    else
        atomicAdd(&d_mes_nbs[i], 1);

    // if (Xi.x < 1.f) {
    //     dF.x = 0.f;
    //     dF.f = 0.f;
    //     dF.w = 0.f;
    // }

    if (Xi.w<0.f) dF.w=0.f;
    if (Xi.f<0.f) dF.f=0.f;
    return dF;
}

__device__ float wall_friction(Cell Xi, Cell r, float dist, int i, int j)
{
    if (i == j) return 0;
    // if (Xi.x < 1.0f) return 0;
    return 1;
}

__device__ void link_force(const Cell* __restrict__ d_X, const int a,
    const int b, const float strength, Cell* d_dX)
{
    if(d_X[a].f + d_X[b].f> 0.2f) return;

    auto r = d_X[a] - d_X[b];
    auto dist = norm3df(r.x, r.y, r.z);

    atomicAdd(&d_dX[a].x, -strength * r.x / dist);
    atomicAdd(&d_dX[a].y, -strength * r.y / dist);
    atomicAdd(&d_dX[a].z, -strength * r.z / dist);
    atomicAdd(&d_dX[b].x, strength * r.x / dist);
    atomicAdd(&d_dX[b].y, strength * r.y / dist);
    atomicAdd(&d_dX[b].z, strength * r.z / dist);
}

__global__ void update_protrusions(const int n_cells,
    const Grid<n_max>* __restrict__ d_grid, const Cell* __restrict d_X,
    hiprandState* d_state, Link* d_link)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells * prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5) / prots_per_cell);
    auto rand_nb_cube =
        d_grid->d_cube_id[j] +
        d_nhood[min(static_cast<int>(hiprand_uniform(&d_state[i]) * 27), 26)];
    auto cells_in_cube =
        d_grid->d_cube_end[rand_nb_cube] - d_grid->d_cube_start[rand_nb_cube];
    if (cells_in_cube < 1) return;

    auto a = d_grid->d_point_id[j];
    auto b =
        d_grid->d_point_id[d_grid->d_cube_start[rand_nb_cube] +
                           min(static_cast<int>(
                                   hiprand_uniform(&d_state[i]) * cells_in_cube),
                               cells_in_cube - 1)];
    D_ASSERT(a >= 0);
    D_ASSERT(a < n_cells);
    D_ASSERT(b >= 0);
    D_ASSERT(b < n_cells);
    if (a == b) return;

    if ((d_type[a] != mesenchyme) or (d_type[b] != mesenchyme)) return;

    auto new_r = d_X[a] - d_X[b];
    auto new_dist = norm3df(new_r.x, new_r.y, new_r.z);
    if (new_dist > r_protrusion) return;

    auto link = &d_link[a * prots_per_cell + i % prots_per_cell];
    auto not_initialized = link->a == link->b;
    auto old_r = d_X[link->a] - d_X[link->b];
    auto old_dist = norm3df(old_r.x, old_r.y, old_r.z);
    auto noise = hiprand_uniform(&d_state[i]);

    auto high_f = false;
    // auto high_f = (d_X[a].f + d_X[b].f) > 0.2f;
    auto distal = (d_X[a].f + d_X[b].f) > 0.025f;//0.025f;//0.20f; //0.025
    bool more_along_w = false;
    bool normal_to_f_gradient = false;
    bool normal_to_w = false;
    if(distal) {
        more_along_w =
            fabs(new_r.w / new_dist) > fabs(old_r.w / old_dist) * (1.f - noise);
        // normal_to_f_gradient =
        //     fabs(new_r.f / new_dist) < fabs(old_r.f / old_dist) * (1.f - noise);
        // normal_to_w =
        //     fabs(new_r.w / new_dist) < fabs(old_r.w / old_dist) * (1.f - noise);
        // high_f = true;
    } else {
        more_along_w =
            fabs(new_r.w / new_dist) > fabs(old_r.w / old_dist) * (1.f - noise);
        // normal_to_f_gradient =
        //     fabs(new_r.f / new_dist) < fabs(old_r.f / old_dist) * (1.f - noise);
        // high_f = true;
    }
    // high_f = false;
    // high_f = true;
    if (not_initialized or more_along_w or high_f or normal_to_f_gradient) {
        link->a = a;
        link->b = b;
    }
}

__global__ void proliferate(float max_rate, float mean_distance, Cell* d_X,
    int* d_n_cells, hiprandState* d_state)
{
    D_ASSERT(*d_n_cells * max_rate <= n_max);
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *d_n_cells * (1 - max_rate))
        return;  // Dividing new cells is problematic!

    // float rate = d_prolif_rate[i] * d_X[i].f;
    // float rate = d_prolif_rate[i] - d_prolif_rate[i]*(1.f - 0.25f)*(1.f-d_X[i].f);
    float rate = d_prolif_rate[i];


    switch (d_type[i]) {
        case mesenchyme: {
            auto r = hiprand_uniform(&d_state[i]);
            if (r > rate) return;
            break;
        }
        // case epithelium: {
        //     // if (d_epi_nbs[i] > d_mes_nbs[i]) return;
        //     if (d_epi_nbs[i] > 7) return;
        //     if (d_mes_nbs[i] <= 0) return;
        //     auto r = hiprand_uniform(&d_state[i]);
        //     if (r > 2.5f * rate) return;  // 2.5
        // }
        default: {
            // if (d_epi_nbs[i] > d_mes_nbs[i]) return;
            if (d_epi_nbs[i] > 7) return;
            if (d_mes_nbs[i] <= 0) return;
            auto r = hiprand_uniform(&d_state[i]);
            if (r > 2.5f * rate) return;  // 2.5
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto theta = hiprand_uniform(&d_state[i]) * 2 * M_PI;
    auto phi = hiprand_uniform(&d_state[i]) * M_PI;
    d_X[n].x = d_X[i].x + mean_distance / 4 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance / 4 * sinf(theta) * sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance / 4 * cosf(theta);
    if (d_type[i] == mesenchyme) {
        d_X[n].w = d_X[i].w / 2;
        d_X[i].w = d_X[i].w / 2;
        d_X[n].f = d_X[i].f / 2;
        d_X[i].f = d_X[i].f / 2;
    } else {
        d_X[n].w = d_X[i].w;
        d_X[n].f = d_X[i].f;
    }
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
    d_prolif_rate[n] = d_prolif_rate[i];
    // d_mes_nbs[n] = 0;
    // d_epi_nbs[n] = 0;
}

// Double step solver means we have to initialise n_neibhbours before every
// step.
// This function is called before each step.
void neighbour_init(const Cell* __restrict__ d_X, Cell* d_dX)
{
    thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + n_max, 0);
    thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_max, 0);
}

template<int n_links, typename Pt = float3,
    Link_force<Pt> force = linear_force<Pt>>
void link_forces_w_n_init(
    Links<n_links>& links, const Pt* __restrict__ d_X, Pt* d_dX)
{
    thrust::fill(thrust::device, n_epi_nbs.d_prop, n_epi_nbs.d_prop + n_max, 0);
    thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_max, 0);
    link<Pt, force><<<(links.get_d_n() + 32 - 1) / 32, 32>>>(
        d_X, d_dX, links.d_link, links.get_d_n(), links.strength);
}

//*****************************************************************************

int main(int argc, char const* argv[])
{
    std::string file_name = argv[1];
    std::string output_tag = argv[2];
    float max_proliferation_rate = std::stof(argv[3]);
    int n_time_steps = std::stoi(argv[4]);
    int prolif_dist = std::stoi(argv[5]);

    // Load the initial conditions
    Vtk_input input(file_name);
    int n0 = input.n_bolls;
    Solution<Cell, n_max, Grid_solver> limb(n0);

    input.read_positions(limb);
    input.read_polarity(limb);

    Property<n_max, Cell_types> type;
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    Property<n_max, int> intype;

    input.read_property(intype, "cell_type");  // we read it as an int, then we translate to
                                               // enum "Cell_types"
    for (int i = 0; i < n0; i++) {
        limb.h_X[i].w = 0.0f;
        limb.h_X[i].f = 0.0f;
        if (intype.h_prop[i] == 0) {
            type.h_prop[i] = mesenchyme;
        } else if (intype.h_prop[i] == 1) {
            type.h_prop[i] = epithelium;
            limb.h_X[i].w = 1.0f;
        } else {
            type.h_prop[i] = aer;
            limb.h_X[i].w = 1.0f;
            limb.h_X[i].f = 1.0f;
        }
    }

    limb.copy_to_device();
    type.copy_to_device();

    std::cout << "initial nbolls " << n0 << " nmax " << n_max << std::endl;

    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

    Links<static_cast<int>(n_max * prots_per_cell)> protrusions(
        protrusion_strength, n0 * prots_per_cell);
    auto intercalation = std::bind(
        link_forces_w_n_init<static_cast<int>(n_max * prots_per_cell), Cell, link_force>,
        protrusions, std::placeholders::_1, std::placeholders::_2);

    Grid<n_max> grid;

    // determine cell-specific proliferation rates
    Property<n_max, float> prolif_rate("prolif_rate");
    hipMemcpyToSymbol(HIP_SYMBOL(
        d_prolif_rate), &prolif_rate.d_prop, sizeof(d_prolif_rate));

    float min_proliferation_rate = 0.5f * max_proliferation_rate;
    // if (prolif_dist == 0) {
        for (int i = 0; i < n0; i++) {
            prolif_rate.h_prop[i] = max_proliferation_rate;
        }
    // } else {
    //     float xmax = -10000.0f;
    //     for (int i = 0; i < n0; i++) {
    //         if (limb.h_X[i].x > xmax) xmax = limb.h_X[i].x;
    //     }
    //     for (int i = 0; i < n0; i++) {
    //         if (limb.h_X[i].x < 0)
    //             prolif_rate.h_prop[i] = 0;
    //         else
    //             prolif_rate.h_prop[i] = min_proliferation_rate +
    //                                     pow((limb.h_X[i].x / xmax), 1) *
    //                                         max_proliferation_rate * 0.5f;
    //     }
    // }
    prolif_rate.copy_to_device();

    // State for proliferations
    hiprandState* d_state;
    hipMalloc(&d_state, n_max * sizeof(hiprandState));
    auto seed = time(NULL);
    setup_rand_states<<<(n_max + 128 - 1) / 128, 128>>>(
        n_max, seed, d_state);

    int skip_step = 1;  // n_time_steps/10;
    std::cout << "n_time_steps " << n_time_steps << " write interval "
              << skip_step << std::endl;

    Vtk_output limb_output(output_tag);

    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        if (time_step % skip_step == 0 || time_step == n_time_steps) {
            limb.copy_to_host();
            protrusions.copy_to_host();
            type.copy_to_host();
            n_epi_nbs.copy_to_host();
            n_mes_nbs.copy_to_host();
            prolif_rate.copy_to_host();
        }

        proliferate<<<(limb.get_d_n() + 128 - 1) / 128, 128>>>(
            max_proliferation_rate, r_min, limb.d_X, limb.d_n, d_state);
        protrusions.set_d_n(limb.get_d_n() * prots_per_cell);
        grid.build(limb, r_protrusion);
        update_protrusions<<<(protrusions.get_d_n() + 32 - 1) / 32, 32>>>(
            limb.get_d_n(), grid.d_grid, limb.d_X, protrusions.d_state,
            protrusions.d_link);

        limb.take_step<wall_force, wall_friction>(dt, intercalation);

        // write the output
        if (time_step % skip_step == 0 || time_step == n_time_steps) {
            limb_output.write_positions(limb);
            limb_output.write_links(protrusions);
            limb_output.write_polarity(limb);
            limb_output.write_field(limb, "Wint");
            limb_output.write_field(limb, "FGF", &Cell::f);
            limb_output.write_property(type);
            limb_output.write_property(n_epi_nbs);
            limb_output.write_property(n_mes_nbs);
            limb_output.write_property(prolif_rate);
        }
    }

    return 0;
}
