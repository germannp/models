#include "hip/hip_runtime.h"
// Simulating a layer.
#include <assert.h>
#include <cmath>
#include <iostream>
#include <stdio.h>

#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"


const float R_MAX = 1;
const float R_MIN = 0.6;
const int N_CELLS = 20;
const int N_TIME_STEPS = 100;
const float DELTA_T = 0.5;


struct pocell {
    float x, y, z, phi, theta;
};

__device__ pocell operator+(const pocell& a, const pocell& b) {
    pocell sum = {a.x + b.x, a.y + b.y, a.z + b.z, a.phi + b.phi, a.theta + b.theta};
    return sum;
}

__device__ pocell operator*(const pocell& a, const float b) {
    pocell prod = {a.x*b, a.y*b, a.z*b, a.phi*b, a.theta*b};
    return prod;
}

__device__ __managed__ Solution<pocell, N_CELLS, LatticeSolver> X;


// Cubic potential plus (p_i . r_ij)^2/2r
__device__ pocell epithelium(pocell Xi, pocell Xj, int i, int j) {
    pocell dF = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    if (i == j) return dF;

    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    float F = 2*(R_MIN - dist)*(R_MAX - dist) + powf(R_MAX - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    // n1 . n2 = sin(t1)*sin(t2)*cos(p1 - p2) + cos(t1)*cos(t2)
    float r_phi = atanf(r.y/r.x);
    float r_theta = acosf(r.z/dist);
    float prod = sinf(Xi.theta)*sinf(r_theta)*cosf(Xi.phi - r_phi) +
        cosf(Xi.theta)*cosf(r_theta);
    dF.phi = prod*(sinf(Xi.theta)*sinf(r_theta)*sinf(Xi.phi - r_phi));
    dF.theta = - prod*(cosf(Xi.theta)*sinf(r_theta)*cosf(Xi.phi - r_phi) -
        sinf(Xi.theta)*cosf(r_theta));

    assert(dF.x == dF.x);  // For NaN f != f.
    return dF;
}

__device__ __managed__ nhoodint<pocell> potential = epithelium;


// Write polarity
class PocellOutput: public VtkOutput {
public:
    using VtkOutput::VtkOutput;
    template<typename Pt, int N_MAX, template<typename, int> class Solver>
    void write_polarity(int n_cells, Solution<Pt, N_MAX, Solver>& X);
};

template<typename Pt, int N_MAX, template<typename, int> class Solver>
void PocellOutput::write_polarity(int n_cells, Solution<Pt, N_MAX, Solver>& X) {
    if (!mWrite) return;

    std::ofstream file(mCurrentFile, std::ios_base::app);
    assert(file.is_open());

    file << "\nPOINT_DATA " << n_cells << "\n";
    file << "NORMALS polarity float\n";
    float3 n = {0, 0, 0};
    for (int i = 0; i < n_cells; i++) {
        n.x = sinf(X[i].theta)*cosf(X[i].phi);
        n.y = sinf(X[i].theta)*sinf(X[i].phi);
        n.z = cosf(X[i].theta);
        file << n.x << " " << n.y << " " << n.z << "\n";
    }
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    // uniform_sphere(N_CELLS, 0.733333, X);
    uniform_circle(N_CELLS, 0.733333/2, X);
    for (int i = 0; i < N_CELLS; i++) {
        X[i].phi = rand()/(RAND_MAX + 1.)*M_PI;
        X[i].theta = rand()/(RAND_MAX + 1.)*2*M_PI;
    }

    // Integrate cell positions
    PocellOutput output("epithelium");
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(N_CELLS, X);
        output.write_polarity(N_CELLS, X);
        if (time_step == N_TIME_STEPS) return 0;

        X.step(DELTA_T, potential);
    }
}
