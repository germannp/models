#include "hip/hip_runtime.h"
// Simulating cell sorting with limited interactions.
#include <assert.h>
#include <cmath>
#include <sys/stat.h>
#include <thrust/sort.h>

#include "../lib/vtk.cuh"
#include "../lib/lattice.cuh"


const float R_MAX = 1;
const float R_MIN = 0.5;
const int N_CELLS = 1000;
const int N_TIME_STEPS = 300;
const float DELTA_T = 0.05;

__device__ __managed__ float3 X[N_CELLS];


__device__ float3 cell_cell_interaction(float3 Xi, float3 Xj) {
    float3 dF = {0.0f, 0.0f, 0.0f};
    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = fminf(sqrtf(r.x*r.x + r.y*r.y + r.z*r.z), R_MAX);
    if (dist > 1e-8) {
        float F = 2*(R_MIN - dist)*(R_MAX - dist) + (R_MAX - dist)*(R_MAX - dist);
        dF.x += r.x*F/dist;
        dF.y += r.y*F/dist;
        dF.z += r.z*F/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    int cell_type[N_CELLS];
    float r_sphere = pow(N_CELLS/0.75, 1./3)*R_MIN/2; // Sphere packing
    for (int i = 0; i < N_CELLS; i++) {
        cell_type[i] = (i < N_CELLS/2) ? 0 : 1;
        float r = r_sphere*pow(rand()/(RAND_MAX + 1.), 1./3);
        float theta = rand()/(RAND_MAX + 1.)*2*M_PI;
        float phi = acos(2.*rand()/(RAND_MAX + 1.) - 1);
        X[i].x = r*sin(theta)*sin(phi);
        X[i].y = r*cos(theta)*sin(phi);
        X[i].z = r*cos(phi);
    }

    // Integrate cell positions
    mkdir("output", 755);
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        char file_name[22];
        sprintf(file_name, "output/sorting_%03i.vtk", time_step);
        write_positions(file_name, N_CELLS, X);
        write_scalars(file_name, N_CELLS, "cell_type", cell_type);

        if (time_step < N_TIME_STEPS) {
            euler_step(DELTA_T, N_CELLS, X);
        }
    }

    return 0;
}
