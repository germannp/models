#include "hip/hip_runtime.h"
// Simulate intercalating cells
#include <assert.h>
#include <cmath>
#include <sys/stat.h>
#include <hiprand/hiprand_kernel.h>

#include "../lib/inits.cuh"
#include "../lib/vtk.cuh"
#include "../lib/n2n.cuh"
// #include "../lib/lattice.cuh"


const float R_MAX = 1;
const float R_MIN = 0.5;
const int N_CELLS = 500;
const int N_CONNECTIONS = 250;
const int N_TIME_STEPS = 1000;
const float DELTA_T = 0.1;

__device__ __managed__ float3 X[N_CELLS], dX[N_CELLS];
__device__ __managed__ int connections[N_CONNECTIONS][2];
__device__ __managed__ hiprandState rand_states[N_CONNECTIONS];

__global__ void setup_rand_states() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N_CELLS) hiprand_init(1337, i, 0, &rand_states[i]);
}


__device__ float3 cell_cell_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF = {0.0f, 0.0f, 0.0f};
    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = fminf(sqrtf(r.x*r.x + r.y*r.y + r.z*r.z), R_MAX);
    if (dist > 1e-7) {
        float F = 2*(R_MIN - dist)*(R_MAX - dist) + (R_MAX - dist)*(R_MAX - dist);
        dF.x = r.x*F/dist;
        dF.y = r.y*F/dist;
        dF.z = r.z*F/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


__global__ void intercalate() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N_CONNECTIONS) {
        float3 Xi = X[connections[i][0]];
        float3 Xj = X[connections[i][1]];
        float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
        float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        X[connections[i][0]].x -= r.x/dist*DELTA_T/5;
        X[connections[i][0]].y -= r.y/dist*DELTA_T/5;
        X[connections[i][0]].z -= r.z/dist*DELTA_T/5;
        X[connections[i][1]].x += r.x/dist*DELTA_T/5;
        X[connections[i][1]].y += r.y/dist*DELTA_T/5;
        X[connections[i][1]].z += r.z/dist*DELTA_T/5;

        int j = (int)(hiprand_uniform(&rand_states[i])*N_CELLS);
        int k = (int)(hiprand_uniform(&rand_states[i])*N_CELLS);
        r = {X[j].x - X[k].x, X[j].y - X[k].y, X[j].z - X[k].z};
        dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        if ((fabs(r.x/dist) < 0.2) && (j != k) && (dist < 2)) {
            connections[i][0] = j;
            connections[i][1] = k;
        }
    }
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(N_CELLS, R_MIN, X);
    setup_rand_states<<<(N_CONNECTIONS + 32 - 1)/32, 32>>>();
    hipDeviceSynchronize();
    int i = 0;
    while (i < N_CONNECTIONS) {
        int j = (int)(rand()/(RAND_MAX + 1.)*N_CELLS);
        int k = (int)(rand()/(RAND_MAX + 1.)*N_CELLS);
        float3 r = {X[j].x - X[k].x, X[j].y - X[k].y, X[j].z - X[k].z};
        float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        if ((fabs(r.x/dist) < 0.2) && (j != k) && (dist < 2)) {
            connections[i][0] = j;
            connections[i][1] = k;
            i++;
        }
    }
    // Integrate cell positions
    VtkOutput output("intercalation");
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(N_CELLS, X);
        output.write_connections(N_CONNECTIONS, connections);
        if (time_step < N_TIME_STEPS) {
            euler_step(DELTA_T, N_CELLS, X, dX);
            intercalate<<<(N_CONNECTIONS + 32 - 1)/32, 32>>>();
            hipDeviceSynchronize();
        }
    }

    return 0;
}
