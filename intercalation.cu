#include "hip/hip_runtime.h"
// Simulate intercalating cells
#include <hiprand/hiprand_kernel.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"


const auto R_MAX = 1.f;
const auto R_MIN = 0.5f;
const auto N_CELLS = 500u;
const auto N_CONNECTIONS = 250u;
const auto N_TIME_STEPS = 1000u;
const auto DELTA_T = 0.2f;

__device__ __managed__ Solution<float3, N_CELLS, LatticeSolver> X;
__device__ __managed__ int connections[N_CONNECTIONS][2];
__device__ hiprandState rand_states[N_CONNECTIONS];


__device__ float3 clipped_cubic(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    auto F = 2*(R_MIN - dist)*(R_MAX - dist) + (R_MAX - dist)*(R_MAX - dist);
    dF = r*F/dist;
    return dF;
}

__device__ __managed__ auto d_potential = clipped_cubic;


__global__ void setup_rand_states() {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N_CELLS) hiprand_init(1337, i, 0, &rand_states[i]);
}

__global__ void intercalate(const float3* __restrict__ X, float3* dX) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N_CONNECTIONS) return;

    auto Xi = X[connections[i][0]];
    auto Xj = X[connections[i][1]];
    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);

    atomicAdd(&dX[connections[i][0]].x, -r.x/dist/5);
    atomicAdd(&dX[connections[i][0]].y, -r.y/dist/5);
    atomicAdd(&dX[connections[i][0]].z, -r.z/dist/5);
    atomicAdd(&dX[connections[i][1]].x, r.x/dist/5);
    atomicAdd(&dX[connections[i][1]].y, r.y/dist/5);
    atomicAdd(&dX[connections[i][1]].z, r.z/dist/5);
}

void intercalation(const float3* __restrict__ X, float3* dX) {
    intercalate<<<(N_CONNECTIONS + 32 - 1)/32, 32>>>(X, dX);
    hipDeviceSynchronize();
}

__global__ void update_connections() {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N_CONNECTIONS) return;

    auto j = static_cast<int>(hiprand_uniform(&rand_states[i])*N_CELLS);
    auto k = static_cast<int>(hiprand_uniform(&rand_states[i])*N_CELLS);
    auto r = X[j] - X[k];
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if ((fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
        connections[i][0] = j;
        connections[i][1] = k;
    }
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(R_MIN, X);
    setup_rand_states<<<(N_CONNECTIONS + 32 - 1)/32, 32>>>();
    hipDeviceSynchronize();
    int i = 0;
    while (i < N_CONNECTIONS) {
        auto j = static_cast<int>(rand()/(RAND_MAX + 1.)*N_CELLS);
        auto k = static_cast<int>(rand()/(RAND_MAX + 1.)*N_CELLS);
        auto r = X[j] - X[k];
        auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        if ((fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
            connections[i][0] = j;
            connections[i][1] = k;
            i++;
        }
    }

    // Integrate cell positions
    VtkOutput output("intercalation");
    for (auto time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(X);
        output.write_connections(connections);
        if (time_step == N_TIME_STEPS) return 0;

        X.step(DELTA_T, d_potential, intercalation);
        update_connections<<<(N_CONNECTIONS + 32 - 1)/32, 32>>>();
        hipDeviceSynchronize();
    }
}
