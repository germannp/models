#include "hip/hip_runtime.h"
// Simulate intercalating cells
#include <functional>
#include <hiprand/hiprand_kernel.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/protrusions.cuh"
#include "../lib/vtk.cuh"


const auto R_MAX = 1.f;
const auto R_MIN = 0.5f;
const auto N_CELLS = 500u;
const auto N_LINKS = 250u;
const auto N_TIME_STEPS = 1000u;
const auto DELTA_T = 0.2f;


__device__ float3 pairwise_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > R_MAX) return dF;

    auto F = 2*(R_MIN - dist)*(R_MAX - dist) + (R_MAX - dist)*(R_MAX - dist);
    dF = r*F/dist;
    return dF;
}

#include "../lib/solvers.cuh"


__global__ void update_links(const float3* __restrict__ d_X, Link* d_link,
        hiprandState* d_state) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= N_LINKS) return;

    auto j = min(static_cast<int>(hiprand_uniform(&d_state[i])*N_CELLS), N_CELLS - 1);
    auto k = min(static_cast<int>(hiprand_uniform(&d_state[i])*N_CELLS), N_CELLS - 1);
    auto r = d_X[j] - d_X[k];
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if ((fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
        d_link[i].a = j;
        d_link[i].b = k;
    }
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<float3, N_CELLS, LatticeSolver> bolls;
    uniform_sphere(R_MIN, bolls);
    Protrusions<N_LINKS> links;
    auto intercalation = std::bind(link_forces<N_LINKS>, links,
        std::placeholders::_1, std::placeholders::_2);
    int i = 0;
    while (i < N_LINKS) {
        auto j = static_cast<int>(rand()/(RAND_MAX + 1.)*N_CELLS);
        auto k = static_cast<int>(rand()/(RAND_MAX + 1.)*N_CELLS);
        auto r = bolls.h_X[j] - bolls.h_X[k];
        auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
        if ((fabs(r.x/dist) < 0.2) and (j != k) and (dist < 2)) {
            links.h_link[i].a = j;
            links.h_link[i].b = k;
            i++;
        }
    }
    links.memcpyHostToDevice();

    // Integrate cell positions
    VtkOutput output("intercalation");
    for (auto time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        bolls.memcpyDeviceToHost();
        links.memcpyDeviceToHost();
        update_links<<<(N_LINKS + 32 - 1)/32, 32>>>(bolls.d_X, links.d_link, links.d_state);
        bolls.step(DELTA_T, intercalation);
        output.write_positions(bolls);
        output.write_protrusions(links);
    }

    return 0;
}
