#include "hip/hip_runtime.h"
// Simulate intercalating cells
#include <functional>
#include <hiprand/hiprand_kernel.h>

#include "../lib/dtypes.cuh"
#include "../lib/inits.cuh"
#include "../lib/links.cuh"
#include "../lib/vtk.cuh"


const auto r_max = 1.f;
const auto r_min = 0.5f;
const auto n_cells = 500u;
const auto prots_per_cell = 1;
const auto n_time_steps = 1000u;
const auto dt = 0.2f;


__device__ float3 pairwise_interaction(float3 Xi, float3 Xj, int i, int j) {
    float3 dF {0};
    if (i == j) return dF;

    auto r = Xi - Xj;
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > r_max) return dF;

    auto F = 2*(r_min - dist)*(r_max - dist) + (r_max - dist)*(r_max - dist);
    dF = r*F/dist;
    return dF;
}

#include "../lib/solvers.cuh"


__global__ void update_protrusions(const float3* __restrict__ d_X, Link* d_link,
        hiprandState* d_state) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= n_cells*prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5)/prots_per_cell);
    auto k = min(static_cast<int>(hiprand_uniform(&d_state[i])*n_cells), n_cells - 1);
    if (j == k) return;

    auto r = d_X[j] - d_X[k];
    auto dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if ((fabs(r.x/dist) < 0.2) and (dist < 2)) {
        d_link[i].a = j;
        d_link[i].b = k;
    }
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    Solution<float3, n_cells, Lattice_solver> bolls;
    uniform_sphere(r_min, bolls);
    Links<n_cells*prots_per_cell> protrusions;
    auto intercalation = std::bind(linear_force<n_cells*prots_per_cell>, protrusions,
        std::placeholders::_1, std::placeholders::_2);

    // Integrate cell positions
    Vtk_output output("intercalation");
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        bolls.copy_to_host();
        protrusions.copy_to_host();
        update_protrusions<<<(n_cells*prots_per_cell + 32 - 1)/32, 32>>>(bolls.d_X,
            protrusions.d_link, protrusions.d_state);
        bolls.take_step(dt, intercalation);
        output.write_positions(bolls);
        output.write_links(protrusions);
    }

    return 0;
}
