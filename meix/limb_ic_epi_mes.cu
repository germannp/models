#include "hip/hip_runtime.h"
// Makes initial conditions for a limb bud taking the morphology from a 3D model
//(3D mesh), then fills the volume with mesenchymal cells and the surface with
// epithelial cells, then lets teh system relax.

// Command line arguments
// argv[1]=input mesh file name
// argv[2]=output file tag
// argv[3]=target limb bud size (dx)
// argv[4]=cube relax_time
// argv[5]=limb bud relax_time
// argv[6]=links flag (activate if you want to use links in later simulations)
// argv[7]=wall flag (activate in limb buds, when you want a wall boundary cond.).
// argv[8]=AER flag (activate in limb buds)

#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <list>
#include <string>
#include <vector>
#include "../../include/dtypes.cuh"
#include "../../include/inits.cuh"
#include "../../include/links.cuh"
#include "../../include/polarity.cuh"
#include "../../include/property.cuh"
#include "../../include/solvers.cuh"
#include "../../include/vtk.cuh"

#include "meix.h"

const auto r_max = 1.0;
const auto r_min = 0.8;
const auto dt = 0.1;
const auto n_max = 150000;
const auto prots_per_cell = 1;
const auto protrusion_strength = 0.2f;
const auto r_protrusion = 2.0f;

enum Cell_types { mesenchyme, epithelium, aer };

__device__ Cell_types* d_type;
__device__ int* d_freeze;

MAKE_PT(Cell, theta, phi);

__device__ Cell relaxation_force(Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    if (i == j) return dF;

    if (d_freeze[i] == 1)
        return dF;  // frozen cells don't experience force so don't move

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        if (d_type[i] == mesenchyme)
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0);
        else
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0) * 2.f;
    } else {
        F = fmaxf(0.9 - dist, 0) * 2.f - fmaxf(dist - 0.9, 0) * 2.f;
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    if (d_type[i] >= epithelium && d_type[j] >= epithelium)
        dF += rigidity_force(Xi, r, dist) * 0.10f;

    return dF;
}

__device__ Cell wall_force(Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    if (i == j) return dF;

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        if (d_type[i] == mesenchyme)
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0);
        else
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0) * 2.f;
    } else {
        F = fmaxf(0.9 - dist, 0) * 2.f - fmaxf(dist - 0.9, 0) * 2.f;
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    if (d_type[i] >= epithelium && d_type[j] >= epithelium)
        dF += rigidity_force(Xi, r, dist) * 0.5f;

    if (Xi.x < 0) dF.x = 0.f;

    return dF;
}

__global__ void update_protrusions(const int n_cells,
    const Grid<n_max>* __restrict__ d_grid, const Cell* __restrict d_X,
    hiprandState* d_state, Link* d_link)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells * prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5) / prots_per_cell);
    auto rand_nb_cube =
        d_grid->d_cube_id[j] +
        d_nhood[min(static_cast<int>(hiprand_uniform(&d_state[i]) * 27), 26)];
    auto cells_in_cube =
        d_grid->d_cube_end[rand_nb_cube] - d_grid->d_cube_start[rand_nb_cube];
    if (cells_in_cube < 1) return;

    auto a = d_grid->d_point_id[j];
    auto b =
        d_grid->d_point_id[d_grid->d_cube_start[rand_nb_cube] +
                           min(static_cast<int>(
                                   hiprand_uniform(&d_state[i]) * cells_in_cube),
                               cells_in_cube - 1)];
    D_ASSERT(a >= 0);
    D_ASSERT(a < n_cells);
    D_ASSERT(b >= 0);
    D_ASSERT(b < n_cells);
    if (a == b) return;

    if ((d_type[a] != mesenchyme) or (d_type[b] != mesenchyme)) return;

    auto new_r = d_X[a] - d_X[b];
    auto new_dist = norm3df(new_r.x, new_r.y, new_r.z);
    if (new_dist > r_protrusion) return;

    auto link = &d_link[a * prots_per_cell + i % prots_per_cell];
    auto not_initialized = link->a == link->b;
    auto new_one = hiprand_uniform(&d_state[i]) < 0.05f;
    if (not_initialized || new_one) {
        link->a = a;
        link->b = b;
    }
}

__device__ float relaxation_friction(Cell Xi, Cell r, float dist, int i, int j)
{
    return 0;
}

__device__ float freeze_friction(Cell Xi, Cell r, float dist, int i, int j)
{
    if (d_freeze[i] == 1) return 0;
    return 1;
}

__device__ float wall_friction(Cell Xi, Cell r, float dist, int i, int j)
{
    if (Xi.x < 0) return 0;
    return 1;
}

// Distribute bolls uniformly random in rectangular cube
template<typename Pt, int n_max, template<typename, int> class Solver>
void uniform_cubic_rectangle(float xmin, float ymin, float zmin, float dx,
    float dy, float dz, Solution<Pt, n_max, Solver>& bolls,
    unsigned int n_0 = 0)
{
    assert(n_0 < *bolls.h_n);

    for (auto i = n_0; i < *bolls.h_n; i++) {
        bolls.h_X[i].x = xmin + dx * (rand() / (RAND_MAX + 1.));
        bolls.h_X[i].y = ymin + dy * (rand() / (RAND_MAX + 1.));
        bolls.h_X[i].z = zmin + dz * (rand() / (RAND_MAX + 1.));
        bolls.h_X[i].phi = 0.0f;
        bolls.h_X[i].theta = 0.0f;
    }

    bolls.copy_to_device();
}

template<typename Pt, int n_max, template<typename, int> class Solver>
void fill_solver_w_meix_no_flank(
    Meix meix, Solution<Pt, n_max, Solver>& bolls, unsigned int n_0 = 0)
{
    // eliminate the flank boundary
    int i = 0;
    while (i < meix.Facets.size()) {
        if (meix.Facets[i].N.x > -1.01f && meix.Facets[i].N.x < -0.99f)
            meix.Facets.erase(meix.Facets.begin() + i);
        else
            i++;
    }
    meix.n_facets = meix.Facets.size();

    *bolls.h_n = meix.n_facets;
    assert(n_0 < *bolls.h_n);

    for (int j = 0; j < meix.n_facets; j++) {
        Triangle T = meix.Facets[j];
        float r = sqrt(pow(T.N.x, 2) + pow(T.N.y, 2) + pow(T.N.z, 2));
        bolls.h_X[j].x = T.C.x;
        bolls.h_X[j].y = T.C.y;
        bolls.h_X[j].z = T.C.z;
        bolls.h_X[j].phi = atan2(T.N.y, T.N.x);
        bolls.h_X[j].theta = acos(T.N.z / r);
    }
}

template<typename Pt, int n_max, template<typename, int> class Solver,
    typename Prop>
void fill_solver_w_epithelium(Solution<Pt, n_max, Solver>& inbolls,
    Solution<Pt, n_max, Solver>& outbolls, Prop& type, unsigned int n_0 = 0)
{
    assert(n_0 < *inbolls.h_n);
    assert(n_0 < *outbolls.h_n);

    int j = 0;
    for (int i = 0; i < *inbolls.h_n; i++) {
        if (type.h_prop[i] == epithelium) {
            outbolls.h_X[j].x = inbolls.h_X[i].x;
            outbolls.h_X[j].y = inbolls.h_X[i].y;
            outbolls.h_X[j].z = inbolls.h_X[i].z;
            outbolls.h_X[j].phi = inbolls.h_X[i].phi;
            outbolls.h_X[j].theta = inbolls.h_X[i].theta;
            j++;
        }
    }
    *outbolls.h_n = j;
}


int main(int argc, char const* argv[])
{

    std::string file_name = argv[1];
    std::string output_tag = argv[2];
    float target_dx = std::stof(argv[3]);
    int cube_relax_time = std::stoi(argv[4]);
    int epi_relax_time = std::stoi(argv[5]);
    bool links_flag = false;
    if(stoi(argv[6]) == 1)
        links_flag = true;
    bool wall_flag = false;
    if(stoi(argv[7]) == 1)
        wall_flag = true;
    bool AER_flag = false;
    if(stoi(argv[8]) == 1)
        AER_flag = true;

    // First, load the mesh file so we can get the maximum dimensions of the
    // system
    Meix meix(file_name);

    // Compute max length in X axis to know how much we need to rescale
    //**********************************************************************
    // Attention! we are assuming the PD axis of the limb is aligned with X
    //**********************************************************************

    float xmin = 10000.0f, xmax = -10000.0f;
    float ymin, ymax, zmin, zmax;
    float dx, dy, dz;

    for (int i = 0; i < meix.n_vertices; i++) {
        if (meix.Vertices[i].x < xmin) xmin = meix.Vertices[i].x;
        if (meix.Vertices[i].x > xmax) xmax = meix.Vertices[i].x;
    }
    dx = xmax - xmin;

    float resc = target_dx / dx;
    std::cout << "xmax= " << xmax << " xmin= " << xmin << std::endl;
    std::cout << "dx= " << dx << " target_dx= " << target_dx
              << " rescaling factor resc= " << resc << std::endl;


    // meix defines the overall shape of the limb bud (mesench. + ectoderm)
    meix.Rescale_relative(resc);
    // meix_mesench defines the volume occupied by the mesenchyme (smaller than
    // meix)
    Meix meix_mesench = meix;
    meix_mesench.Rescale_absolute(-r_min, wall_flag);  //*1.3//*1.2
    // Compute min. and max, positions in x,y,z from rescaled mesh
    xmin = 10000.0f;
    xmax = -10000.0f;
    ymin = 10000.0f;
    ymax = -10000.0f;
    zmin = 10000.0f;
    zmax = -10000.0f;
    for (int i = 0; i < meix_mesench.n_vertices; i++) {
        if (meix_mesench.Vertices[i].x < xmin)
            xmin = meix_mesench.Vertices[i].x;
        if (meix_mesench.Vertices[i].x > xmax)
            xmax = meix_mesench.Vertices[i].x;
        if (meix_mesench.Vertices[i].y < ymin)
            ymin = meix_mesench.Vertices[i].y;
        if (meix_mesench.Vertices[i].y > ymax)
            ymax = meix_mesench.Vertices[i].y;
        if (meix_mesench.Vertices[i].z < zmin)
            zmin = meix_mesench.Vertices[i].z;
        if (meix_mesench.Vertices[i].z > zmax)
            zmax = meix_mesench.Vertices[i].z;
    }
    dx = xmax - xmin;
    dy = ymax - ymin;
    dz = zmax - zmin;

    // we use the maximum lengths of the mesh to draw a cube that includes the
    // mesh
    // Let's fill the cube with bolls

    // Now we include intercalation in the cubic relaxation, so we must assume a
    // larger cube, since the end result will be compressed to some extent
    float factor = 0.1f;
    float r = dx * factor / 2;
    float new_xmin = xmin - r;
    r = dy * factor / 2;
    float new_ymin = ymin - r;
    r = dz * factor / 2;
    float new_zmin = zmin - r;
    float new_dx = dx + dx * factor, new_dy = dy + dy * factor,
          new_dz = dz + dz * factor;

    float cube_vol = new_dx * new_dy * new_dz;
    float r_boll = 0.5f * r_min;
    float boll_vol = 4.f / 3.f * M_PI * pow(r_boll, 3);
    int n_bolls_cube = cube_vol / boll_vol;

    std::cout << "cube dims " << dx << " " << dy << " " << dz << std::endl;
    std::cout << "cube_vol " << cube_vol << std::endl;
    std::cout << "r_boll " << r_boll << std::endl;
    std::cout << "boll_vol " << boll_vol << std::endl;
    std::cout << "nbolls in cube " << n_bolls_cube << std::endl;

    Solution<Cell, n_max, Grid_solver> cube(n_bolls_cube);
    // Fill the cube with bolls
    uniform_cubic_rectangle(
        new_xmin, new_ymin, new_zmin, new_dx, new_dy, new_dz, cube);

    // Variable indicating cell type
    Property<n_max, Cell_types> type;
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    // Variable that indicates which cells are 'frozen', so don't move
    Property<n_max, int> freeze("freeze");
    hipMemcpyToSymbol(HIP_SYMBOL(d_freeze), &freeze.d_prop, sizeof(d_freeze));

    for (auto i = 0; i < n_bolls_cube; i++) {
        type.h_prop[i] = mesenchyme;
        freeze.h_prop[i] = 0;
    }

    cube.copy_to_device();
    type.copy_to_device();
    freeze.copy_to_device();

    // Declaration of links
    Links<static_cast<int>(n_max * prots_per_cell)> protrusions(
        protrusion_strength, n_bolls_cube * prots_per_cell);
    auto intercalation =
    std::bind(link_forces<static_cast<int>(n_max * prots_per_cell), Cell>,
        protrusions, std::placeholders::_1, std::placeholders::_2);

    Grid<n_max> grid;

    // State for links
    hiprandState* d_state;
    hipMalloc(&d_state, n_max * sizeof(hiprandState));
    setup_rand_states<<<(n_max + 128 - 1) / 128, 128>>>(d_state, n_max);

    // Relaxation of the cube
    int skip_step = 1;  // relax_time/10;
    // std::cout<<"relax_time "<<relax_time<<" write interval "<<
    // skip_step<<std::endl;

    // Vtk_output cubic_output1(output_tag+".cubic_relaxation1");

    for (auto time_step = 0; time_step <= cube_relax_time; time_step++) {
        // if(time_step%skip_step==0 || time_step==cube_relax_time){
        //     cube.copy_to_host();
        // }

        cube.take_step<relaxation_force, relaxation_friction>(dt);

        // write the output
        // if(time_step%skip_step==0 || time_step==cube_relax_time) {
        //     cubic_output1.write_positions(cube);
        // }
    }

    std::cout<<"Cube 1 integrated"<<std::endl;

    // The relaxed cube positions will be used to imprint epithelial cells
    cube.copy_to_host();
    std::vector<Point> cube_relax_points;
    for (auto i = 0; i < n_bolls_cube; i++) {
        Point p = Point(cube.h_X[i].x, cube.h_X[i].y, cube.h_X[i].z);
        cube_relax_points.push_back(p);
    }

    if(links_flag) {

        // Vtk_output cubic_output(output_tag+".cubic_relaxation");

        // We apply the links to the relaxed cube to compress it (as will be the
        // mesench in the limb bud)
        for (auto time_step = 0; time_step <= cube_relax_time; time_step++) {
            // if(time_step%skip_step==0 || time_step==cube_relax_time){
            //     cube.copy_to_host();
            //     protrusions.copy_to_host();
            // }

            protrusions.set_d_n(cube.get_d_n() * prots_per_cell);
            grid.build(cube, r_protrusion);
            update_protrusions<<<(protrusions.get_d_n() + 32 - 1) / 32, 32>>>(
                cube.get_d_n(), grid.d_grid, cube.d_X, protrusions.d_state,
                protrusions.d_link);

            cube.take_step<relaxation_force, relaxation_friction>(
                dt, intercalation);

            // write the output
            // if(time_step%skip_step==0 || time_step==cube_relax_time) {
            //     cubic_output.write_positions(cube);
            //     cubic_output.write_links(protrusions);
            // }
        }
        std::cout
            <<"Cube 2 integrated with links (only when links flag is active)"
            <<std::endl;
    }

    // Fit the cube into a mesh and sort which cells are inside the mesh
    // For the mesenchyme we use the smaller mesh and the compressed cube
    // For the epithelium we use the larger meix and the relaxed cube

    // Mesenchyme
    // Setup the list of points
    std::vector<Point> cube_points;
    for (auto i = 0; i < n_bolls_cube; i++) {
        Point p = Point(cube.h_X[i].x, cube.h_X[i].y, cube.h_X[i].z);
        cube_points.push_back(p);
    }

    // Setup the list of inclusion test results
    int* mesench_result = new int[n_bolls_cube];
    // Set direction of ray
    Point dir = Point(0.0f, 1.0f, 0.0f);

    meix_mesench.InclusionTest(cube_points, mesench_result, dir);

    // Make a new list with the ones that are inside
    std::vector<Point> mes_cells;
    int n_bolls_mes = 0;
    for (int i = 0; i < n_bolls_cube; i++) {
        if (mesench_result[i] == 1) {
            mes_cells.push_back(cube_points[i]);
            n_bolls_mes++;
        }
    }

    std::cout << "bolls_in_cube " << n_bolls_cube << " bolls after fill "
              << n_bolls_mes << std::endl;

    // Epithelium (we have to sort out which ones are inside the big mesh and
    // out of the small one)
    // Setup the list of inclusion test results
    int* epi_result_big = new int[n_bolls_cube];
    int* epi_result_small = new int[n_bolls_cube];

    meix.InclusionTest(cube_relax_points, epi_result_big, dir);
    meix_mesench.InclusionTest(cube_relax_points, epi_result_small, dir);

    // Make a new list with the ones that are inside
    std::vector<Point> epi_cells;
    int n_bolls_epi = 0;
    for (int i = 0; i < n_bolls_cube; i++) {
        if (epi_result_big[i] == 1 and epi_result_small[i] == 0) {
            epi_cells.push_back(cube_relax_points[i]);
            n_bolls_epi++;
        }
    }

    int n_bolls_total = n_bolls_mes + n_bolls_epi;

    std::cout << "bolls_in_mes " << n_bolls_mes << " bolls_in_epi "
              << n_bolls_epi << " bolls_in_total " << n_bolls_total
              << std::endl;

    Solution<Cell, n_max, Grid_solver> bolls(n_bolls_total);

    for (int i = 0; i < n_bolls_mes; i++) {
        bolls.h_X[i].x = mes_cells[i].x;
        bolls.h_X[i].y = mes_cells[i].y;
        bolls.h_X[i].z = mes_cells[i].z;
        type.h_prop[i] = mesenchyme;
        freeze.h_prop[i] = 1;
    }
    int count = 0;
    for (int i = n_bolls_mes; i < n_bolls_total; i++) {
        bolls.h_X[i].x = epi_cells[count].x;
        bolls.h_X[i].y = epi_cells[count].y;
        bolls.h_X[i].z = epi_cells[count].z;
        type.h_prop[i] = epithelium;
        freeze.h_prop[i] = 0;
        // polarity
        Point p = epi_cells[count];
        int f = -1;
        float dmin = 1000000.f;
        // we use the closest facet on meix to determine the polarity of the
        // epithelial cell
        for (int j = 0; j < meix.n_facets; j++) {
            Point r = p - meix.Facets[j].C;
            float d = sqrt(r.x * r.x + r.y * r.y + r.z * r.z);
            if (d < dmin) {
                dmin = d;
                f = j;
            }
        }
        count++;
        if (meix.Facets[f].C.x < 0.1f && wall_flag) {  // the cells contacting the flank
                                                       // boundary can't be epithelial 0.001
            type.h_prop[i] = mesenchyme;
            freeze.h_prop[i] = 1;
            continue;
        }
        bolls.h_X[i].phi = atan2(meix.Facets[f].N.y, meix.Facets[f].N.x);
        bolls.h_X[i].theta = acos(meix.Facets[f].N.z);
    }
    std::cout << "count " << count << " epi_cells " << n_bolls_epi << std::endl;

    bolls.copy_to_device();
    type.copy_to_device();
    freeze.copy_to_device();

    std::cout << "n_bolls_total= " << n_bolls_total << std::endl;

    if(AER_flag) {
        // Imprint the AER on the epithelium (based on a mesh file too)
        std::string AER_file=file_name;
        AER_file.insert(AER_file.length() - 4, "_AER");
        std::cout<<"AER file "<<AER_file<<std::endl;
        Meix AER(AER_file);
        AER.Rescale_relative(resc);

        for (int i = n_bolls_mes; i < n_bolls_total; i++) {
            Point p(bolls.h_X[i].x, bolls.h_X[i].y, bolls.h_X[i].z);
            for (int j = 0; j < AER.n_facets; j++) {
                Point r = p - AER.Facets[j].C;
                float d = sqrt(r.x * r.x + r.y * r.y + r.z * r.z);
                if (d < r_min*1.5f) {
                    type.h_prop[i]=aer;
                    break;
                }
            }
        }

        AER.WriteVtk(output_tag + ".aer");
    }

    Vtk_output output(output_tag);

    skip_step = 1;  // relax_time/10;
    for (auto time_step = 0; time_step <= epi_relax_time; time_step++) {
        if (time_step % skip_step == 0 || time_step == epi_relax_time) {
            bolls.copy_to_host();
        }

        bolls.take_step<relaxation_force, freeze_friction>(dt);

        // write the output
        if (time_step % skip_step == 0 || time_step == epi_relax_time) {
            output.write_positions(bolls);
            output.write_polarity(bolls);
            output.write_property(type);
            output.write_property(freeze);
        }
    }

    // bolls.copy_to_device();
    // output.write_positions(bolls);
    // output.write_polarity(bolls);
    // output.write_property(type);

    // Unfreeze the mesenchyme
    for (int i = 0; i < n_bolls_total; i++) {
        if (type.h_prop[i] == mesenchyme) freeze.h_prop[i] = 0;
    }

    // write down the meix in the vtk file to compare it with the posterior
    // seeding
    meix.WriteVtk(output_tag);
    // write down the mesenchymal mesh in the vtk file to compare it with the
    // posterior filling
    meix_mesench.WriteVtk(output_tag + ".mesench");

    // Create a dummy meix that depicts the x=0 plane, depicting the flank
    // boundary
    Meix wall;
    Point A(0.f, 2 * ymin, 2 * zmin);
    Point B(0.f, 2 * ymin, 2 * zmax);
    Point C(0.f, 2 * ymax, 2 * zmin);
    Point D(0.f, 2 * ymax, 2 * zmax);
    Point N(1.f, 0.f, 0.f);
    Triangle ABC(A, B, C, N);
    Triangle BCD(B, C, D, N);
    wall.n_facets = 2;
    wall.Facets.push_back(ABC);
    wall.Facets.push_back(BCD);
    wall.WriteVtk(output_tag + ".wall");

    // for shape comparison purposes we write down the initial mesh as the
    // facets
    // centres and the bolls epithelium in separate vtk files.

    std::cout << "writing meix_T0" << std::endl;
    Solution<Cell, n_max, Grid_solver> meix_T0(meix.n_facets);
    fill_solver_w_meix_no_flank(meix, meix_T0);
    Vtk_output output_meix_T0(output_tag + ".meix_T0");
    output_meix_T0.write_positions(meix_T0);
    output_meix_T0.write_polarity(meix_T0);

    std::cout << "writing epi_T0" << std::endl;
    Solution<Cell, n_max, Grid_solver> epi_T0(n_bolls_total);
    fill_solver_w_epithelium(bolls, epi_T0, type);
    Vtk_output output_epi_T0(output_tag + ".epi_T0");
    output_epi_T0.write_positions(epi_T0);
    output_epi_T0.write_polarity(epi_T0);

    std::cout << "DOOOOOOOOOOOOOOONE***************" << std::endl;

    return 0;
}
