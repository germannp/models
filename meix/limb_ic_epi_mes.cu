#include "hip/hip_runtime.h"
//Makes initial conditions for a limb bud taking the morphology from a 3D model
//(3D mesh), then fills the volume with mesenchymal cells and the surface with
//epithelial cells, then lets teh system relax.

#include "../../include/dtypes.cuh"
#include "../../include/inits.cuh"
#include "../../include/solvers.cuh"
#include "../../include/vtk.cuh"
#include "../../include/polarity.cuh"
#include "../../include/property.cuh"
#include <sstream>
#include <string>
#include <list>
#include <vector>
#include <iostream>

#include "meix.h"

const auto r_max=1.0;
const auto r_min=0.6;

const auto dt = 0.05*r_min*r_min;

const auto n_0 = 1000;
const auto n_max = 65000;

enum Cell_types {mesenchyme, epithelium};

__device__ Cell_types* d_type;
 // __device__ int* d_mes_nbs;  // number of mesenchymal neighbours
 // __device__ int* d_epi_nbs;

//__device__ Cell_types* d_cube_type;


MAKE_PT(Cell, x, y, z, theta, phi);

__device__ Cell relaxation_force(Cell Xi, Cell Xj, int i, int j) {
    Cell dF {0};

    if(i==j) return dF;

    //FOR TESTING PURPOSES ONLY //we use epithelium type to reproduce the mesh within the bolls framework
    //if(d_type[i]==epithelium || d_type[j]==epithelium) return dF;
    //*************************//

    auto r = Xi - Xj;
    auto dist = norm3df(r.x, r.y, r.z);
    if (dist > r_max) return dF;

    auto F = 2*(r_min - dist)*(r_max - dist) + powf(r_max - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    if(d_type[i]==epithelium && d_type[j]==epithelium)
    {
      dF += rigidity_force(Xi, Xj)*0.2;//*3;
    }

    // if (d_type[j] == epithelium) {atomicAdd(&d_epi_nbs[i],1);}
    // else {atomicAdd(&d_mes_nbs[i],1);}

    return dF;
}

// Distribute bolls uniformly random in rectangular cube
template<typename Pt, int n_max, template<typename, int> class Solver>
void uniform_cubic_rectangle(float x0,float y0,float z0,float dx,float dy,float dz, Solution<Pt, n_max, Solver>& bolls, unsigned int n_0 = 0)
{

    assert(n_0 < *bolls.h_n);

    for (auto i = n_0; i < *bolls.h_n; i++) {
      //std::cout<<"crash5.1.3 "<< i<<std::endl;
        bolls.h_X[i].x = x0+dx*(rand()/(RAND_MAX+1.));
        bolls.h_X[i].y = y0+dy*(rand()/(RAND_MAX+1.));
        bolls.h_X[i].z = z0+dz*(rand()/(RAND_MAX+1.));
        bolls.h_X[i].phi=0.0f;
        bolls.h_X[i].theta=0.0f;

    }

    bolls.copy_to_device();
}

//this function will distribute uniformly random epithelial cells on top of the
//mesh surface.
void seed_epithelium_on_meix(Meix& meix, std::vector<Cell>& cells, float n_epi)
{

//std::cout<<"crash0.1"<<std::endl;
    int nF=meix.Facets.size();
    //std::cout<<"nF "<<nF<<std::endl;
    for(int i=0; i< n_epi; i++)
    {
      int j=rand()%nF; //which facet will fall into

      Point V0= meix.Facets[j].V0;
      Point V1= meix.Facets[j].V1;
      Point V2= meix.Facets[j].V2;
      Point N= meix.Facets[j].N;

      float phi=atan2(N.y,N.x);
      float theta= acos(N.z);

      bool bingo=false;
      while (!bingo)
      {
        // 0<s<1 ; 0<t<1 ; s+t<1
        float s=rand()/(RAND_MAX+1.f);
        float t=rand()/(RAND_MAX+1.f);
        if (s+t>1) continue;

        float a=1-s-t;

        Point p= V0*a + V1*s + V2*t;

        Cell c;

        c.x = p.x;
        c.y = p.y;
        c.z = p.z;
        c.phi = phi;
        c.theta = theta;
        cells.push_back(c);
//std::cout<<i<<" crash0.4 j= "<<j<<" c= "<<c<<" n= "<<n<<std::endl;
        bingo=true;
      }
    }
}

//*****************************************************************************

template<typename Pt, int n_max, template<typename, int> class Solver, typename Prop>
void epithelium_mesenchyme_assembly(std::vector<Point>& mes_cells, std::vector<Cell>& epi_cells, Property<n_max, Prop>& type, Solution<Pt, n_max, Solver>& bolls, unsigned int n_0 = 0)
{
  assert(n_0 < *bolls.h_n);
  int n_mes=mes_cells.size();
  int n_epi=epi_cells.size();

  for (int i=0 ; i<n_mes ; i++)
  {
    bolls.h_X[i].x = mes_cells[i].x;
    bolls.h_X[i].y = mes_cells[i].y;
    bolls.h_X[i].z = mes_cells[i].z;
    bolls.h_X[i].theta = 0.f;
    bolls.h_X[i].phi = 0.f;
    type.h_prop[i]=mesenchyme;
  }

  for (int i=0 ; i<n_epi ; i++)
  {
    bolls.h_X[n_mes+i].x = epi_cells[i].x;
    bolls.h_X[n_mes+i].y = epi_cells[i].y;
    bolls.h_X[n_mes+i].z = epi_cells[i].z;
    bolls.h_X[n_mes+i].theta = epi_cells[i].theta;
    bolls.h_X[n_mes+i].phi = epi_cells[i].phi;
    type.h_prop[i]=epithelium;
  }

  bolls.copy_to_device();
  type.copy_to_device();

}

//writes the whole meix data structure as a vtk file
void write_meix_vtk (Meix meix, std::string output_tag)
{
  std::string filename="output/"+output_tag+".meix.vtk";
  std::ofstream meix_file(filename);
  assert(meix_file.is_open());

  int n=meix.Facets.size();

  meix_file << "# vtk DataFile Version 3.0\n";
  meix_file << output_tag+".meix" << "\n";
  meix_file << "ASCII\n";
  meix_file << "DATASET POLYDATA\n";

  meix_file << "\nPOINTS " << 3*n << " float\n";
  for (auto i = 0; i < n; i++)
  {
    meix_file <<meix.Facets[i].V0.x << " " << meix.Facets[i].V0.y << " " << meix.Facets[i].V0.z << "\n";
    meix_file <<meix.Facets[i].V1.x << " " << meix.Facets[i].V1.y << " " << meix.Facets[i].V1.z << "\n";
    meix_file <<meix.Facets[i].V2.x << " " << meix.Facets[i].V2.y << " " << meix.Facets[i].V2.z << "\n";
  }

  meix_file << "\nPOLYGONS " << n << " " << 4*n << "\n";
  for (auto i = 0; i < 3*n; i+=3)
  {
    meix_file << "3 " << i <<" "<<i+1 <<" "<<i+2 << "\n";
  }
  meix_file.close();

}

int main(int argc, char const *argv[])
{

  // Command line arguments
  // argv[1]=output file tag
  // argv[2]=mesh file name
  // argv[3]=target limb bud size (dx)
  // argv[4]=cube relax_time
  // argv[5]=assumed cell radius
  // argv[6]=post assembly relax time

  std::string output_tag=argv[1];
  std::string file_name=argv[2];

  //First, load the mesh file so we can get the maximum dimensions of the system
  Meix meix(file_name);

  //Compute max length in X axis to know how much we need to rescale
  //**********************************************************************
  //Attention! we are assuming the PD axis of the limb is aligned with X
  //**********************************************************************

  float xmin=10000.0f,xmax=-10000.0f;
  float ymin,ymax,zmin,zmax;
  float dx,dy,dz;

  for(int i=0 ; i<meix.Facets.size() ; i++)
  {
    if(meix.Facets[i].C.x<xmin) xmin=meix.Facets[i].C.x;  if(meix.Facets[i].C.x>xmax) xmax=meix.Facets[i].C.x;
  }
  dx=xmax-xmin;

  float target_dx=std::stof(argv[3]);
  float resc=target_dx/dx;
  std::cout<<"xmax= "<<xmax<<" xmin= "<<xmin<<std::endl;
  std::cout<<"dx= "<<dx<<" target_dx= "<<target_dx<<" rescaling factor resc= "<<resc<<std::endl;

  meix.Rescale(resc);

  //Compute min. and max, positions in x,y,z from rescaled mesh
  xmin=10000.0f;xmax=-10000.0f;ymin=10000.0f;ymax=-10000.0f;zmin=10000.0f;zmax=-10000.0f;
  for(int i=0 ; i<meix.Facets.size() ; i++)
  {
    if(meix.Facets[i].C.x<xmin) xmin=meix.Facets[i].C.x;  if(meix.Facets[i].C.x>xmax) xmax=meix.Facets[i].C.x;
    if(meix.Facets[i].C.y<ymin) ymin=meix.Facets[i].C.y;  if(meix.Facets[i].C.y>ymax) ymax=meix.Facets[i].C.y;
    if(meix.Facets[i].C.z<zmin) zmin=meix.Facets[i].C.z;  if(meix.Facets[i].C.z>zmax) zmax=meix.Facets[i].C.z;
  }
  dx=xmax-xmin;
  dy=ymax-ymin;
  dz=zmax-zmin;

  //we use the maximum lengths of the mesh to draw a cube that includes the mesh
  //Let's fill the cube with bolls
  //How many bolls? We calculate the volume of the cube we want to fill
  //then we calculate how many bolls add up to that volume, correcting by the
  //inefficiency of a cubic packing (0.74)----> Well in the end we don't correct cause it wasn't packed enough

  //const float packing_factor=0.74048f;
  float cube_vol=dx*dy*dz;
  float r_boll=0.3f;
  float boll_vol=4./3.*M_PI*pow(r_boll,3);
  int n_bolls_cube=cube_vol/boll_vol;

  std::cout<<"cube dims "<<dx<<" "<<dy<<" "<<dz<<std::endl;
  std::cout<<"nbolls in cube "<<n_bolls_cube<<std::endl;

  Solution<Cell, n_max, Lattice_solver> cube(n_bolls_cube);
  //Fill the rectangle with bolls
  uniform_cubic_rectangle(xmin,ymin,zmin,dx,dy,dz,cube);

  Property<n_max, Cell_types> type;
  hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
  for (auto i = 0; i < n_bolls_cube; i++)
  {
      type.h_prop[i] = mesenchyme;
  }

  cube.copy_to_device();
  type.copy_to_device();
  // Property<n_max, int> n_mes_nbs;
  // hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
  // Property<n_max, int> n_epi_nbs;
  // hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

  // We run the solver on bolls so the cube of bolls relaxes
  std::stringstream ass;
  ass << argv[1] << ".cubic_relaxation";
  std::string cubic_out = ass.str();

  int relax_time=std::stoi(argv[4]);
  int write_interval=relax_time/10;
  std::cout<<"relax_time "<<relax_time<<" write interval "<< write_interval<<std::endl;

  Vtk_output cubic_output(cubic_out);
  // std::cout<<"crash0"<<std::endl;
  for (auto time_step = 0; time_step <= relax_time; time_step++)
  {
    // std::cout<<"crash1"<<std::endl;
    if(time_step%write_interval==0 || time_step==relax_time)
    {
      // std::cout<<"crash2"<<std::endl;
      cube.copy_to_host();
    }
// std::cout<<"crash3"<<std::endl;
    cube.build_lattice(r_max);
// std::cout<<"crash4"<<std::endl;
    // thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_bolls_cube, 0);

    cube.take_step<relaxation_force>(dt);
// std::cout<<"crash5"<<std::endl;
    //write the output
    if(time_step%write_interval==0 || time_step==relax_time)
    {
      // std::cout<<"crash6"<<std::endl;
      cubic_output.write_positions(cube);
      cubic_output.write_polarity(cube);
      // std::cout<<"crash7"<<std::endl;
    }

  }


  //Find the bolls that are inside the mesh and store their positions
  //METHOD: Shooting a ray from a ball and counting how many triangles intersects.
  //If the ray intersects an even number of facets the boll is out of the mesh, else is in

  //Setup the list of points
  std::vector<Point> points;
  for (auto i = 0; i < n_bolls_cube; i++)
  {
    Point p=Point(cube.h_X[i].x, cube.h_X[i].y, cube.h_X[i].z);
    points.push_back(p);
  }

  //Setup the list of inclusion test results
  int* results=new int[n_bolls_cube];
  //Set direction of ray
  Point dir=Point(1.0f,0.0f,0.0f);

  meix.InclusionTest(points , results, dir);

  //Make a new list with the ones that are inside
  std::vector<Point> mes_cells;
  int n_bolls_mes=0;
  for (int i = 0; i < n_bolls_cube; i++)
  {
    if(results[i]==1)
    {
      mes_cells.push_back(points[i]);
      n_bolls_mes++;
    }
  }

  std::cout<<"bolls_in_cube "<<n_bolls_cube<<" bolls after fill "<<n_bolls_mes<<std::endl;


  //The mesenchyme is done, now we make the epithelium

  //In order to efficiently seed the mesh surface, we need to estimate epithelial
  //cell density based on the cell size. We have cell radius (bolls eq. radius)
  //as input from cmd. line. We assume hexagonal packing of epithelia, so the
  //effective surface occupied by one cell will be the one of an hexagon with
  //apothem equal to the cell radius

  float cell_radius=std::stof(argv[5]);
  float cell_S=cell_radius*cell_radius*6.f/sqrt(3.f); //regular hexagon formula
  float cell_density=1.f/cell_S;

  //Calculate whole Surface area of meix
  int n_bolls_epi=std::round(cell_density*meix.SurfArea);

  std::cout<<"nbolls_epi= "<<n_bolls_epi<<" cell_density= "<<cell_density<<" meix_S= "<<meix.SurfArea<<std::endl;

  std::vector<Cell> epi_cells;

  //seed the cells onto the meix
  seed_epithelium_on_meix(meix, epi_cells,n_bolls_epi);

  int n_bolls_total=n_bolls_mes+n_bolls_epi;
  Solution<Cell, n_max, Lattice_solver> bolls(n_bolls_total);
  // Property<n_max, Cell_types> type;
  // hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));

  epithelium_mesenchyme_assembly(mes_cells, epi_cells, type,bolls);

  // for (int i = 0; i < n_bolls; i++)
  // {
  //     type.h_prop[i] = epithelium;
  // }
  // bolls.copy_to_device();
  // type.copy_to_device();

  // Property<n_max, int> n_mes_nbs;
  // hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
  // Property<n_max, int> n_epi_nbs;
  // hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));


Vtk_output output(output_tag);

relax_time=std::stoi(argv[6]);
write_interval=relax_time/10;

for (auto time_step = 0; time_step <= relax_time; time_step++)
{
  if(time_step%write_interval==0 || time_step==relax_time)
  {
    bolls.copy_to_host();
  }

  bolls.build_lattice(r_max);

  //thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_0, 0);

  bolls.take_step<relaxation_force>(dt);

  //write the output
  if(time_step%write_interval==0 || time_step==relax_time)
  {
    output.write_positions(bolls);
    output.write_polarity(bolls);
    output.write_property(type);
  }

}

  //write down the meix in the vtk file to compare it with the posterior seeding
  write_meix_vtk(meix,output_tag);

  return 0;
}
