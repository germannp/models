#include "hip/hip_runtime.h"
//Takes the mesenchyme tissue (as bolls) fitted from a limb 3D model (mesh)
//and makes limb bud initial conditions ready to feed into a model of limb morphogenesis.
//This implies:
//- adding an epithelial layer surrounding the mesenchyme.----DONE
//- making the tissue stable while being as close as possible to the original shape
//- Marking the boundary conditions (i.e. the surface that contacts the body flank)

//TODO
//- making the tissue stable while being as close as possible to the original shape
//      Set the initial polarities of epithelial cells normal to the surface
//      --> Load original mesh and take its normals.
//- Marking the boundary conditions (i.e. the surface that contacts the body flank)
//      Suggestion: the boundary surface is flat, all normals point in the same direction...

//Eventually ic_from_mesh.cu should be fused with this program for simplicity,
//however the former is slow, since it involves simulation. For building and
//testing this part I can use the one set of output files from the ic_from_mesh.cu


#include "../../include/dtypes.cuh"
#include "../../include/solvers.cuh"
#include "../../include/vtk.cuh"
#include "../../include/polarity.cuh"
#include "../../include/property.cuh"

using namespace std;

MAKE_PT(Cell, x, y, z, theta, phi);

const int relax_time=1000;
const auto n_max = 65000;
const auto r_max=1.0;
const auto r_min=0.6;

const auto dt = 0.05*r_min*r_min;

enum Cell_types {mesenchyme, epithelium};

__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;  // number of mesenchymal neighbours
__device__ int* d_epi_nbs;
//__device__ float* d_r_mean;


__device__ Cell relaxation_force(Cell Xi, Cell Xj, int i, int j) {
    Cell dF {0};

    if (i == j ) return dF;

    auto r = Xi - Xj;
    auto dist = norm3df(r.x, r.y, r.z);
    if (dist > r_max) return dF;


    auto F = 2*(r_min - dist)*(r_max - dist) + powf(r_max - dist, 2);
    dF.x = r.x*F/dist;
    dF.y = r.y*F/dist;
    dF.z = r.z*F/dist;

    //diffusion only takes place in epithelium
    if(d_type[i]==epithelium && d_type[j]==epithelium)
    {
      dF += rigidity_force(Xi, Xj)*0.1;//*3;
    }

    if (d_type[j] == epithelium) {atomicAdd(&d_epi_nbs[i],1);}
    else {atomicAdd(&d_mes_nbs[i],1);}

    return dF;
}



int main (int argc, char const *argv[])
{

  //Command-line arguments
  //argv[1]: input file tag
  //argv[2]: output file tag


  //string filename=argv[2];
  //figure out the file paths
  string input_tag=argv[1];
  string output_tag=argv[2];


  string filename="output/"+input_tag+"_1.vtk";
  string meix_filename="output/"+input_tag+".meix_1.vtk";

  std::cout<<"mesenchyme input file "<<filename<<std::endl;
  std::cout<<"mesh input file "<<meix_filename<<std::endl;

  //We load the fitted mesenchyme into a Solution
  int n_bolls;
  Vtk_input input_bolls(filename,n_bolls);
  Solution<Cell, n_max, Lattice_solver> bolls(n_bolls);
  input_bolls.read_positions(bolls);
  input_bolls.read_polarity(bolls);

  Property<n_max, Cell_types> type;
  hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
  for (auto i = 0; i < n_bolls; i++) {
      type.h_prop[i] = mesenchyme;
  }
  bolls.copy_to_device();
  type.copy_to_device();
  Property<n_max, int> n_mes_nbs;
  hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
  Property<n_max, int> n_epi_nbs;
  hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

  //We load the original mesh that has been transformed into a "bolls" epithelium
  //by "ic_from_mesh.cu". We just need the facet centroids (stored as cell positions)
  //and the facet normals (stored as cell polarities).
  int n_meix;
  Vtk_input input_meix(meix_filename,n_meix);
  Solution<Cell, n_max, Lattice_solver> meix(n_meix);
  input_meix.read_positions(meix);
  input_meix.read_polarity(meix);


  //done reading //now we'll add some cell types, epithelium on the surface, mesenchyme inside


  // we need to compute 1 iteration to compute the number of neighbours for each cell
  for (auto time_step = 0; time_step <= 1; time_step++) {
      bolls.build_lattice(r_max);

      // update_protrusions<<<(protrusions.get_d_n() + 32 - 1)/32, 32>>>(bolls.d_lattice,
      //     bolls.d_X, bolls.get_d_n(), protrusions.d_link, protrusions.d_state);
      thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_bolls, 0);
      // bolls.take_step<lb_force>(dt, intercalation);

      bolls.take_step<relaxation_force>(dt);
  }

  // Find epithelium
  bolls.copy_to_host();
  n_mes_nbs.copy_to_host();

  for (auto i = 0; i < n_bolls; i++) {
      if (n_mes_nbs.h_prop[i] < 25) {
        type.h_prop[i] = epithelium;
        auto dist = sqrtf(bolls.h_X[i].x*bolls.h_X[i].x
            + bolls.h_X[i].y*bolls.h_X[i].y + bolls.h_X[i].z*bolls.h_X[i].z);

        //Now, for each epithelial cell, we find the nearest facet from the meix
        //and take its normal
        int closest;
        float min_dist=10000000.f;
        Cell diff;
        for(int j=0 ; j<n_meix ; j++)
        {
          diff=bolls.h_X[i]-meix.h_X[j];
          float d=sqrt(pow(diff.x,2)+pow(diff.y,2)+pow(diff.z,2));
          if(d < min_dist)
          {closest=j; min_dist=d;}
        }

        bolls.h_X[i].theta = meix.h_X[closest].theta;
        bolls.h_X[i].phi = meix.h_X[closest].phi;

      } else {
          bolls.h_X[i].theta = 0;
          bolls.h_X[i].phi = 0;
      }
      // bolls.h_X[i].w = 0;
      // bolls.h_X[i].f = 0;
  }
  bolls.copy_to_device();
  type.copy_to_device();

  // Relax again to let epithelium stabilise
  std::string out_name=output_tag+"_ic";
  Vtk_output output(out_name);
  for (auto time_step = 0; time_step <= relax_time; time_step++) {
      bolls.copy_to_host();
      bolls.build_lattice(r_max);

      // update_protrusions<<<(protrusions.get_d_n() + 32 - 1)/32, 32>>>(bolls.d_lattice,
      //     bolls.d_X, bolls.get_d_n(), protrusions.d_link, protrusions.d_state);
      thrust::fill(thrust::device, n_mes_nbs.d_prop, n_mes_nbs.d_prop + n_bolls, 0);
      // bolls.take_step<lb_force>(dt, intercalation);

      bolls.take_step<relaxation_force>(dt);

      output.write_positions(bolls);
      output.write_polarity(bolls);
      output.write_property(type);

  }

  return 0;

}
