#include "hip/hip_runtime.h"
// Simulating cell sorting with Leonard-Jones potential.
#include <assert.h>
#include <cmath>
#include <sys/stat.h>
#include <hiprand/hiprand_kernel.h>

#include "../lib/sphere.cuh"
#include "../lib/vtk.cuh"
#include "../lib/n2n.cuh"


const float R_MIN = 0.5;
const float MIN_DIST = 0.35;
const int N_CELLS = 200;
const int N_TIME_STEPS = 50000;
const int SKIP_STEPS = 100;
const float DELTA_T = 0.0001;

__device__ __managed__ float3 X[N_CELLS];
__device__ __managed__ hiprandState rand_states[N_CELLS];


__device__ float3 cell_cell_interaction(float3 Xi, float3 Xj, int i, int j) {
    int strength = (1 + 2*(j < N_CELLS/2))*(1 + 2*(i < N_CELLS/2));
    float3 dF = {0.0f, 0.0f, 0.0f};
    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    if (dist > 1e-7) {
        dist = fmaxf(dist, MIN_DIST);
        float r_rel = R_MIN/dist;
        float F = powf(r_rel, 13);
        F -= powf(r_rel, 7);
        F += hiprand_normal(&rand_states[i])*10/sqrtf(N_CELLS);
        dF.x = strength*r.x*F/dist;
        dF.y = strength*r.y*F/dist;
        dF.z = strength*r.z*F/dist;
    }
    assert(dF.x == dF.x); // For NaN f != f.
    return dF;
}


__global__ void setup_rand_states() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N_CELLS) hiprand_init(1337, i, 0, &rand_states[i]);
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(N_CELLS, R_MIN, X);
    int cell_type[N_CELLS];
    for (int i = 0; i < N_CELLS; i++) {
        cell_type[i] = (i < N_CELLS/2) ? 0 : 1;
    }
    setup_rand_states<<<(N_CELLS + 32 - 1)/32, 32>>>();
    hipDeviceSynchronize();

    // Integrate cell positions
    VtkOutput output("sorting-lj", SKIP_STEPS);
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(N_CELLS, X);
        output.write_field(N_CELLS, "cell_type", cell_type);

        if (time_step < N_TIME_STEPS) {
            euler_step(DELTA_T, N_CELLS, X);
        }
    }

    return 0;
}
