#include "hip/hip_runtime.h"
// Simulating cell sorting with Leonard-Jones potential.
#include <assert.h>
#include <cmath>
#include <hiprand/hiprand_kernel.h>

#include "../lib/inits.cuh"
#include "../lib/solvers.cuh"
#include "../lib/vtk.cuh"


const float R_MIN = 0.5;
const float MIN_DIST = 0.35;
const int N_CELLS = 200;
const int N_TIME_STEPS = 50000;
const int SKIP_STEPS = 100;
const float DELTA_T = 0.0001;

__device__ __managed__ Solution<float3, N_CELLS, N2nSolver> X;
__device__ __managed__ hiprandState rand_states[N_CELLS];


__device__ float3 lj_sorting(float3 Xi, float3 Xj, int i, int j) {
    float3 dF = {0.0f, 0.0f, 0.0f};
    if (i == j) return dF;

    int strength = (1 + 2*(j < N_CELLS/2))*(1 + 2*(i < N_CELLS/2));
    float3 r = {Xi.x - Xj.x, Xi.y - Xj.y, Xi.z - Xj.z};
    float dist = sqrtf(r.x*r.x + r.y*r.y + r.z*r.z);
    dist = fmaxf(dist, MIN_DIST);
    float r_rel = R_MIN/dist;
    float F = powf(r_rel, 13);
    F -= powf(r_rel, 7);
    F += hiprand_normal(&rand_states[i])*10/sqrtf(N_CELLS);
    dF.x = strength*r.x*F/dist;
    dF.y = strength*r.y*F/dist;
    dF.z = strength*r.z*F/dist;
    assert(dF.x == dF.x);  // For NaN f != f.
    return dF;
}

__device__ __managed__ nhoodint<float3> p_sorting = lj_sorting;


__global__ void setup_rand_states() {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < N_CELLS) hiprand_init(1337, i, 0, &rand_states[i]);
}


int main(int argc, char const *argv[]) {
    // Prepare initial state
    uniform_sphere(N_CELLS, R_MIN, X);
    int cell_type[N_CELLS];
    for (int i = 0; i < N_CELLS; i++) {
        cell_type[i] = (i < N_CELLS/2) ? 0 : 1;
    }
    setup_rand_states<<<(N_CELLS + 32 - 1)/32, 32>>>();
    hipDeviceSynchronize();

    // Integrate cell positions
    VtkOutput output("sorting-lj", N_TIME_STEPS, SKIP_STEPS);
    for (int time_step = 0; time_step <= N_TIME_STEPS; time_step++) {
        output.write_positions(N_CELLS, X);
        output.write_type(N_CELLS, cell_type);
        if (time_step == N_TIME_STEPS) return 0;

        X.step(DELTA_T, p_sorting);
    }
}
